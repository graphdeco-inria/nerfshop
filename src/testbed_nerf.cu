#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   testbed_nerf.cu
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/adam_optimizer.h>
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/common_nerf.h>
#include <neural-graphics-primitives/envmap.cuh>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/nerf_network.h>
#include <neural-graphics-primitives/marching_cubes.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>

#include <neural-graphics-primitives/editing/tools/affine_bounding_box.cuh>
#include <neural-graphics-primitives/editing/cage_deformation.h>
#include <neural-graphics-primitives/editing/tools/selection_utils.h>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/loss.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/optimizer.h>
#include <tiny-cuda-nn/trainer.h>

#include <filesystem/directory.h>
#include <filesystem/path.h>

#include <queue>

// DEBUG: used for chrono timing 
#include <chrono>

#ifdef copysign
#undef copysign
#endif

using namespace Eigen;
using namespace tcnn;
namespace fs = filesystem;

NGP_NAMESPACE_BEGIN

static constexpr uint32_t MARCH_ITER = 10000;

static constexpr uint32_t MIN_STEPS_INBETWEEN_COMPACTION = 1;
static constexpr uint32_t MAX_STEPS_INBETWEEN_COMPACTION = 8;

struct AuxRay
{
	bool emptying;
	Ray ray;
	float cone_angle;
	float max_level;
	float startt;
	Vector3f light_dir_warped;
	bool max_level_rand_training;
	uint32_t ignoring;
	uint32_t ray_index;
};

struct AuxSample
{
	bool overwritten;
	bool inside;
	bool emptying;
	bool outside;
};

struct LossAndGradient {
	Eigen::Array3f loss;
	Eigen::Array3f gradient;

	__host__ __device__ LossAndGradient operator*(float scalar) {
		return {loss * scalar, gradient * scalar};
	}

	__host__ __device__ LossAndGradient operator/(float scalar) {
		return {loss / scalar, gradient / scalar};
	}
};

inline __device__ Array3f copysign(const Array3f& a, const Array3f& b) {
	return {
		copysignf(a.x(), b.x()),
		copysignf(a.y(), b.y()),
		copysignf(a.z(), b.z()),
	};
}

inline __device__ LossAndGradient l2_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	return {
		difference * difference,
		2.0f * difference
	};
}

inline __device__ LossAndGradient relative_l2_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	Array3f factor = (prediction * prediction + Array3f::Constant(1e-2f)).inverse();
	return {
		difference * difference * factor,
		2.0f * difference * factor
	};
}

inline __device__ LossAndGradient l1_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	return {
		difference.abs(),
		copysign(Array3f::Ones(), difference),
	};
}

inline __device__ LossAndGradient huber_loss(const Array3f& target, const Array3f& prediction, float alpha = 1) {
	Array3f difference = prediction - target;
	Array3f abs_diff = difference.abs();
	Array3f square = 0.5f/alpha * difference * difference;
	return {
		{
			abs_diff.x() > alpha ? (abs_diff.x() - 0.5f * alpha) : square.x(),
			abs_diff.y() > alpha ? (abs_diff.y() - 0.5f * alpha) : square.y(),
			abs_diff.z() > alpha ? (abs_diff.z() - 0.5f * alpha) : square.z(),
		},
		{
			abs_diff.x() > alpha ? (difference.x() > 0 ? 1.0f : -1.0f) : (difference.x() / alpha),
			abs_diff.y() > alpha ? (difference.y() > 0 ? 1.0f : -1.0f) : (difference.y() / alpha),
			abs_diff.z() > alpha ? (difference.z() > 0 ? 1.0f : -1.0f) : (difference.z() / alpha),
		},
	};
}

inline __device__ LossAndGradient log_l1_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	Array3f divisor = difference.abs() + Array3f::Ones();
	return {
		divisor.log(),
		copysign(divisor.inverse(), difference),
	};
}

inline __device__ LossAndGradient smape_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	Array3f factor = (0.5f * (prediction.abs() + target.abs()) + Array3f::Constant(1e-2f)).inverse();
	return {
		difference.abs() * factor,
		copysign(factor, difference),
	};
}

inline __device__ LossAndGradient mape_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	Array3f factor = (prediction.abs() + Array3f::Constant(1e-2f)).inverse();
	return {
		difference.abs() * factor,
		copysign(factor, difference),
	};
}




//__device__ Vector3f interpolate_tet2(Eigen::Vector3f coord, const CageDeformation::MiniGPUInfo& info, bool warp)
//{
//	Eigen::Vector3f unwarped_pos = coord;
//	if (warp)
//		unwarped_pos = unwarp_position(coord, info.aabb);
//
//	// Test the bounding box first
//	if (info.original_bbox.contains_base(coord))
//	{
//		int level = mip_from_pos(unwarped_pos);
//
//		uint32_t cell_idx = level * NERF_GRIDVOLUME() + cascaded_grid_idx_at(unwarped_pos, level);
//
//		// If cell contains a triangle, get it(/them)
//		for (uint32_t j = info.original_tet_lut_offsets[cell_idx]; j < info.original_tet_lut_offsets[cell_idx + 1]; j++) {
//			uint32_t tet_idx = info.original_tet_lut_idx[j];
//			// If point is actually in the selected tet
//			if (point_in_tet<float, Eigen::Vector3f>(info.original_vertices[info.tets[4 * tet_idx]], info.original_vertices[info.tets[4 * tet_idx + 1]], info.original_vertices[info.tets[4 * tet_idx + 2]], info.original_vertices[info.tets[4 * tet_idx + 3]], unwarped_pos))
//			{
//				// Compute barycentric coordinates
//				Eigen::Vector4f bary_coord = bary_tet(info.original_vertices[info.tets[4 * tet_idx]], info.original_vertices[info.tets[4 * tet_idx + 1]], info.original_vertices[info.tets[4 * tet_idx + 2]], info.original_vertices[info.tets[4 * tet_idx + 3]], unwarped_pos);
//
//				coord = bary_coord.x() * info.vertices[info.tets[4 * tet_idx]]
//					+ bary_coord.y() * info.vertices[info.tets[4 * tet_idx + 1]]
//					+ bary_coord.z() * info.vertices[info.tets[4 * tet_idx + 2]]
//					+ bary_coord.w() * info.vertices[info.tets[4 * tet_idx + 3]];
//
//				if (warp)
//					coord = warp_position(coord, info.aabb);
//
//				break;
//			}
//		}
//	}
//	return coord;
//}
//
//
//__device__ bool in_source(Eigen::Vector3f& coord, const CageDeformation::MiniGPUInfo& info, bool warp)
//{
//	Eigen::Vector3f unwarped_pos = coord;
//	if (warp)
//		unwarped_pos = unwarp_position(coord, info.aabb);
//
//	if (info.original_bbox.contains_base(coord))
//	{
//		int level = mip_from_pos(unwarped_pos);
//		uint32_t cell_idx = level * NERF_GRIDVOLUME() + cascaded_grid_idx_at(unwarped_pos, level);
//
//		// If cell contains a triangle, get it(/them)
//		for (uint32_t j = info.original_tet_lut_offsets[cell_idx]; j < info.original_tet_lut_offsets[cell_idx + 1]; j++) {
//			uint32_t tet_idx = info.original_tet_lut_idx[j];
//			// If point is actually in the selected tet
//			if (point_in_tet<float, Eigen::Vector3f>(info.original_vertices[info.tets[4 * tet_idx]], info.original_vertices[info.tets[4 * tet_idx + 1]], info.original_vertices[info.tets[4 * tet_idx + 2]], info.original_vertices[info.tets[4 * tet_idx + 3]], unwarped_pos))
//			{
//				return true;
//			}
//		}
//	}
//	return false;
//}
//
//
//__device__ bool in_target(Eigen::Vector3f& coord, const CageDeformation::MiniGPUInfo& info, bool warp)
//{
//	Eigen::Vector3f unwarped_pos = coord;
//	if (warp)
//		unwarped_pos = unwarp_position(coord, info.aabb);
//
//	if (info.bbox.contains_base(coord))
//	{
//		int level = mip_from_pos(unwarped_pos);
//		uint32_t cell_idx = level * NERF_GRIDVOLUME() + cascaded_grid_idx_at(unwarped_pos, level);
//
//		// If cell contains a triangle, get it(/them)
//		for (uint32_t j = info.tet_lut_offsets[cell_idx]; j < info.tet_lut_offsets[cell_idx + 1]; j++) {
//			uint32_t tet_idx = info.tet_lut_idx[j];
//			// If point is actually in the selected tet
//			if (point_in_tet<float, Eigen::Vector3f>(info.vertices[info.tets[4 * tet_idx]], info.vertices[info.tets[4 * tet_idx + 1]], info.vertices[info.tets[4 * tet_idx + 2]], info.vertices[info.tets[4 * tet_idx + 3]], unwarped_pos))
//			{
//				return true;
//			}
//		}
//	}
//	return false;
//}


__device__ void recListIgnored(bool empty_prop, bool* is_emptying, uint32_t* unique_combos, uint32_t& unique_length, Vector3f pos, Distiller** cage_infos, uint32_t edits_offset, uint32_t num_cages, uint32_t ignored)
{
	for (int i = edits_offset; i < num_cages; i++)
	{
		if (cage_infos[i]->in_source(pos, false))
		{
			recListIgnored(empty_prop, is_emptying, unique_combos, unique_length, cage_infos[i]->map(pos, false), cage_infos, i + 1, num_cages, ignored);
			ignored |= (1 << i);
			bool found = false;
			for (int j = 0; j < unique_length; j++)
			{
				if (unique_combos[j] == ignored)
				{
					found = true;
					break;
				}
			}
			empty_prop |= cage_infos[i]->emptying;
			if (!found)
			{
				unique_combos[unique_length] = ignored;
				is_emptying[unique_length] = empty_prop;
				unique_length++;
				if (unique_length >= 20)
					__trap();
			}
		}
		else if (cage_infos[i]->in_target(pos, false)) // Mapping is broken, stop looking
			break;
	}
}


__device__ float network_to_rgb_derivative(float val, ENerfActivation activation) {
	switch (activation) {
		case ENerfActivation::None: return 1.0f;
		case ENerfActivation::ReLU: return val > 0.0f ? 1.0f : 0.0f;
		case ENerfActivation::Logistic: { float density = tcnn::logistic(val); return density * (1 - density); };
		case ENerfActivation::Exponential: return __expf(tcnn::clamp(val, -10.0f, 10.0f));
		default: assert(false);
	}
	return 0.0f;
}

__device__ float network_to_density_derivative(float val, ENerfActivation activation) {
	switch (activation) {
		case ENerfActivation::None: return 1.0f;
		case ENerfActivation::ReLU: return val > 0.0f ? 1.0f : 0.0f;
		case ENerfActivation::Logistic: { float density = tcnn::logistic(val); return density * (1 - density); };
		case ENerfActivation::Exponential: return __expf(tcnn::clamp(val, -15.0f, 15.0f));
		default: assert(false);
	}
	return 0.0f;
}

__device__ Vector3f unwarp_position_derivative(const Vector3f& pos, const BoundingBox& aabb) {
	// return {logit(pos.x()) + 0.5f, logit(pos.y()) + 0.5f, logit(pos.z()) + 0.5f};
	// return pos;

	return aabb.diag();
}

__device__ Vector3f warp_position_derivative(const Vector3f& pos, const BoundingBox& aabb) {
	return unwarp_position_derivative(pos, aabb).cwiseInverse();
}

__device__ Vector3f warp_direction_derivative(const Vector3f& dir) {
	return Vector3f::Constant(0.5f);
}

__device__ Vector3f unwarp_direction_derivative(const Vector3f& dir) {
	return Vector3f::Constant(2.0f);
}

__global__ void extract_srgb_with_activation(const uint32_t n_elements,	const uint32_t rgb_stride, const float* __restrict__ rgbd, float* __restrict__ rgb, ENerfActivation rgb_activation, bool from_linear) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	const uint32_t elem_idx = i / 3;
	const uint32_t dim_idx = i - elem_idx * 3;

	float c = network_to_rgb(rgbd[elem_idx*4 + dim_idx], rgb_activation);
	if (from_linear) {
		c = linear_to_srgb(c);
	}

	rgb[elem_idx*rgb_stride + dim_idx] = c;
}

__global__ void mark_untrained_density_grid(const uint32_t n_elements,  float* __restrict__ grid_out,
	const uint32_t n_training_images,
	const TrainingImageMetadata* __restrict__ metadata,
	const TrainingXForm* training_xforms,
	Vector2i resolution,
	bool clear_visible_voxels
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint32_t level = i / (NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE());
	uint32_t pos_idx = i % (NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE());

	uint32_t x = tcnn::morton3D_invert(pos_idx>>0);
	uint32_t y = tcnn::morton3D_invert(pos_idx>>1);
	uint32_t z = tcnn::morton3D_invert(pos_idx>>2);

	float half_resx=resolution.x()*0.5f;
	float half_resy=resolution.y()*0.5f;

	Vector3f pos = ((Vector3f{(float)x+0.5f, (float)y+0.5f, (float)z+0.5f}) / NERF_GRIDSIZE() - Vector3f::Constant(0.5f)) * scalbnf(1.0f, level) + Vector3f::Constant(0.5f);
	float voxel_radius = 0.5f*SQRT3()*scalbnf(1.0f, level) / NERF_GRIDSIZE();
	int count=0;
	for (uint32_t j=0; j < n_training_images; ++j) {
		if (metadata[j].camera_distortion.mode == ECameraDistortionMode::FTheta) {
			// not supported for now
			count++;
			break;
		}
		Matrix<float, 3, 4> xform = training_xforms[j].start;
		Vector3f ploc = pos - xform.col(3);
		float x = ploc.dot(xform.col(0));
		float y = ploc.dot(xform.col(1));
		float z = ploc.dot(xform.col(2));
		if (z > 0.f) {
			auto focal = metadata[j].focal_length;
			// TODO - add a box / plane intersection to stop thomas from murdering me
			if (fabsf(x) - voxel_radius < z / focal.x() * half_resx && fabsf(y) - voxel_radius < z / focal.y() * half_resy) {
				count++;
				if (count > 0) break;
			}
		}
	}

	if (clear_visible_voxels || (grid_out[i] < 0) != (count <= 0)) {
		grid_out[i] = (count > 0) ? 0.f : -1.f;
	}
}

__global__ void generate_grid_samples_nerf_uniform(Eigen::Vector3i res_3d, const uint32_t step, BoundingBox render_aabb, BoundingBox train_aabb, NerfPosition* __restrict__ out) {
	// check grid_in for negative values -> must be negative on output
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	uint32_t z = threadIdx.z + blockIdx.z * blockDim.z;
	if (x>=res_3d.x() || y>=res_3d.y() || z>=res_3d.z())
		return;
	uint32_t i = x+ y*res_3d.x() + z*res_3d.x()*res_3d.y();
	Vector3f pos = Array3f{(float)x, (float)y, (float)z} * Array3f{1.f/res_3d.x(),1.f/res_3d.y(),1.f/res_3d.z()};
	pos = pos.cwiseProduct(render_aabb.max - render_aabb.min) + render_aabb.min;
	out[i] = { warp_position(pos, train_aabb), warp_dt(MIN_CONE_STEPSIZE()) };
}

// generate samples for uniform grid including constant ray direction
__global__ void generate_grid_samples_nerf_uniform_dir(Eigen::Vector3i res_3d, const uint32_t step, BoundingBox render_aabb, BoundingBox train_aabb, Eigen::Vector3f ray_dir, NerfCoordinate* __restrict__ network_input) {
	// check grid_in for negative values -> must be negative on output
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	uint32_t z = threadIdx.z + blockIdx.z * blockDim.z;
	if (x>=res_3d.x() || y>=res_3d.y() || z>=res_3d.z())
		return;
	uint32_t i = x+ y*res_3d.x() + z*res_3d.x()*res_3d.y();
	Vector3f pos = Array3f{(float)x, (float)y, (float)z} * Array3f{1.f/res_3d.x(),1.f/res_3d.y(),1.f/res_3d.z()};
	pos = pos.cwiseProduct(render_aabb.max - render_aabb.min) + render_aabb.min;
	network_input[i] = { warp_position(pos, train_aabb), warp_direction(ray_dir), warp_dt(MIN_CONE_STEPSIZE()) };
}

__global__ void splat_grid_samples_nerf_max_nearest_neighbor(const uint32_t n_elements, const uint32_t* __restrict__ indices, const tcnn::network_precision_t* network_output, float* __restrict__ grid_out, ENerfActivation rgb_activation, ENerfActivation density_activation) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint32_t local_idx = indices[i];

	// Current setting: optical thickness of the smallest possible stepsize.
	// Uncomment for:   optical thickness of the ~expected step size when the observer is in the middle of the scene
	uint32_t level = 0;//local_idx / (NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE());

	float mlp = network_to_density(float(network_output[i]), density_activation);
	float optical_thickness = mlp * scalbnf(MIN_CONE_STEPSIZE(), level);

	// Positive floats are monotonically ordered when their bit pattern is interpretes as uint.
	// uint atomicMax is thus perfectly acceptable.
	atomicMax((uint32_t*)&grid_out[local_idx], __float_as_uint(optical_thickness));
}

__global__ void splat_grid_samples_nerf_max_nearest_neighbor_already_activated(const uint32_t n_elements, const uint32_t* __restrict__ indices, const tcnn::network_precision_t* network_output, float* __restrict__ grid_out) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint32_t local_idx = indices[i];

	// Current setting: optical thickness of the smallest possible stepsize.
	// Uncomment for:   optical thickness of the ~expected step size when the observer is in the middle of the scene
	uint32_t level = 0;//local_idx / (NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE());

	float mlp = float(network_output[i]);
	float optical_thickness = mlp * scalbnf(MIN_CONE_STEPSIZE(), level);

	// Positive floats are monotonically ordered when their bit pattern is interpretes as uint.
	// uint atomicMax is thus perfectly acceptable.
	atomicMax((uint32_t*)&grid_out[local_idx], __float_as_uint(optical_thickness));
}

__global__ void grid_samples_half_to_float(const uint32_t n_elements, BoundingBox aabb, float* dst, const tcnn::network_precision_t* network_output, ENerfActivation density_activation, const NerfPosition* __restrict__ coords_in, const float* __restrict__ grid_in) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	// let's interpolate for marching cubes based on the raw MLP output, not the density (exponentiated) version
	//float mlp = network_to_density(float(network_output[i * padded_output_width]), density_activation);
	float mlp = float(network_output[i]);

	if (grid_in) {
		Vector3f pos = unwarp_position(coords_in[i].p, aabb);
		float grid_density = cascaded_grid_at(pos, grid_in, mip_from_pos(pos));
		if (grid_density < NERF_MIN_OPTICAL_THICKNESS()) {
			mlp = -10000.f;
		}
	}
	dst[i] = mlp;
}

__global__ void ema_grid_samples_nerf(const uint32_t n_elements,
	float decay,
	const uint32_t count,
	float* __restrict__ grid_out,
	const float* __restrict__ grid_in
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	float importance = grid_in[i];

	// float ema_debias_old = 1 - (float)powf(decay, count);
	// float ema_debias_new = 1 - (float)powf(decay, count+1);

	// float filtered_val = ((grid_out[i] * decay * ema_debias_old + importance * (1 - decay)) / ema_debias_new);
	// grid_out[i] = filtered_val;

	// Maximum instead of EMA allows capture of very thin features.
	// Basically, we want the grid cell turned on as soon as _ANYTHING_ visible is in there.

	float prev_val = grid_out[i];
	float val = (prev_val<0.f) ? prev_val : fmaxf(prev_val * decay, importance);
	grid_out[i] = val;
}

__global__ void decay_sharpness_grid_nerf(const uint32_t n_elements, float decay, float* __restrict__ grid) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;
	grid[i] *= decay;
}

__global__ void grid_to_bitfield(const uint32_t n_elements,
	const float* __restrict__ grid,
	uint8_t* __restrict__ grid_bitfield,
	const float* __restrict__ mean_density_ptr
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint8_t bits = 0;

	float thresh = std::min(NERF_MIN_OPTICAL_THICKNESS(), *mean_density_ptr);

	#pragma unroll
	for (uint8_t j = 0; j < 8; ++j) {
		bits |= grid[i*8+j] > thresh ? ((uint8_t)1 << j) : 0;
	}

	grid_bitfield[i] = bits;
}

__global__ void bitfield_max_pool(const uint32_t n_elements,
	const uint8_t* __restrict__ prev_level,
	uint8_t* __restrict__ next_level
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint8_t bits = 0;

	#pragma unroll
	for (uint8_t j = 0; j < 8; ++j) {
		// If any bit is set in the previous level, set this
		// level's bit. (Max pooling.)
		bits |= prev_level[i*8+j] > 0 ? ((uint8_t)1 << j) : 0;
	}

	uint32_t x = tcnn::morton3D_invert(i>>0) + NERF_GRIDSIZE()/8;
	uint32_t y = tcnn::morton3D_invert(i>>1) + NERF_GRIDSIZE()/8;
	uint32_t z = tcnn::morton3D_invert(i>>2) + NERF_GRIDSIZE()/8;

	next_level[tcnn::morton3D(x, y, z)] |= bits;
}

__global__ void advance_pos_nerf(
	const uint32_t n_elements,
	BoundingBox render_aabb,
	Vector3f camera_fwd,
	Vector2f focal_length,
	uint32_t sample_index,
	NerfPayload* __restrict__ payloads,
	const uint8_t* __restrict__ density_grid,
	uint32_t min_mip,
	float cone_angle_constant
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	NerfPayload& payload = payloads[i];

	if (!payload.alive) {
		return;
	}

	Vector3f origin = payload.origin;
	Vector3f dir = payload.dir;
	Vector3f idir = dir.cwiseInverse();

	float cone_angle = calc_cone_angle(dir.dot(camera_fwd), focal_length, cone_angle_constant);

	float t = payload.t;
	float dt = calc_dt(t, cone_angle);
	t += ld_random_val(sample_index, i * 786433) * dt;
	Vector3f pos;

	while (1) {
		if (!render_aabb.contains(pos = origin + dir * t)) {
			payload.alive = false;
			break;
		}

		dt = calc_dt(t, cone_angle);
		uint32_t mip = max(min_mip, mip_from_dt(dt, pos));

		if (!density_grid || density_grid_occupied_at(pos, density_grid, mip)) {
			break;
		}

		uint32_t res = NERF_GRIDSIZE()>>mip;
		t = advance_to_next_voxel(t, cone_angle, pos, dir, idir, res);
	}

	payload.t = t;
}

__global__ void generate_nerf_network_inputs_from_positions(const uint32_t n_elements, BoundingBox aabb, const Vector3f* __restrict__ pos, PitchedPtr<NerfCoordinate> network_input, Eigen::Vector3f light_dir) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	Vector3f dir=(pos[i]-Vector3f::Constant(0.5f)).normalized(); // choose outward pointing directions, for want of a better choice
	network_input(i)->set_with_optional_light_dir(warp_position(pos[i], aabb), warp_direction(dir), warp_dt(MIN_CONE_STEPSIZE()), warp_direction(light_dir), network_input.stride_in_bytes);
}

__global__ void generate_nerf_network_inputs_at_current_position(const uint32_t n_elements, BoundingBox aabb, const NerfPayload* __restrict__ payloads, PitchedPtr<NerfCoordinate> network_input, Eigen::Vector3f light_dir) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	Vector3f dir = payloads[i].dir;
	network_input(i)->set_with_optional_light_dir(warp_position(payloads[i].origin + dir * payloads[i].t, aabb), warp_direction(dir), warp_dt(MIN_CONE_STEPSIZE()), warp_direction(light_dir), network_input.stride_in_bytes);
}

__global__ void compute_nerf_density(const uint32_t n_elements, Array4f* network_output, ENerfActivation rgb_activation, ENerfActivation density_activation) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	Array4f rgba = network_output[i];
	rgba.w() = tcnn::clamp(1.f - __expf(-network_to_density(rgba.w(), density_activation) / 100.0f), 0.0f, 1.0f);
	rgba.x() = network_to_rgb(rgba.x(), rgb_activation) * rgba.w();
	rgba.y() = network_to_rgb(rgba.y(), rgb_activation) * rgba.w();
	rgba.z() = network_to_rgb(rgba.z(), rgb_activation) * rgba.w();

	network_output[i] = rgba;
}

__global__ void generate_next_nerf_network_inputs(
	const uint32_t n_elements,
	BoundingBox render_aabb,
	BoundingBox train_aabb,
	Vector2f focal_length,
	Vector3f camera_fwd,
	NerfPayload* __restrict__ payloads,
	PitchedPtr<NerfCoordinate> network_input,
	uint32_t n_steps,
	const uint8_t* __restrict__ density_grid,
	uint32_t min_mip,
	float cone_angle_constant,
	Eigen::Vector3f light_dir
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	NerfPayload& payload = payloads[i];

	if (!payload.alive) {
		return;
	}

	Vector3f origin = payload.origin;
	Vector3f dir = payload.dir;
	Vector3f idir = dir.cwiseInverse();

	float cone_angle = calc_cone_angle(dir.dot(camera_fwd), focal_length, cone_angle_constant);

	float t = payload.t;

	for (uint32_t j = 0; j < n_steps; ++j) {
		Vector3f pos;
		float dt = 0.0f;
		while (1) {
			pos = origin + dir * t;
			if (!render_aabb.contains(pos)) {
				payload.n_steps = j;
				return;
			}


			dt = calc_dt(t, cone_angle);
			uint32_t mip = max(min_mip, mip_from_dt(dt, pos));

			if (!density_grid || density_grid_occupied_at(pos, density_grid, mip)) {
				break;
			}

			uint32_t res = NERF_GRIDSIZE()>>mip;
			t = advance_to_next_voxel(t, cone_angle, pos, dir, idir, res);
		}

		network_input(i + j * n_elements)->set_with_optional_light_dir(warp_position(pos, train_aabb), warp_direction(dir), warp_dt(dt), warp_direction(light_dir), network_input.stride_in_bytes); // XXXCONE
		t += dt;
	}

	payload.t = t;
	payload.n_steps = n_steps;
}

__global__ void composite_kernel_nerf(
	const uint32_t n_elements,
	const uint32_t stride,
	const uint32_t current_step,
	BoundingBox aabb,
	float glow_y_cutoff,
	int glow_mode,
	const uint32_t n_training_images,
	const TrainingXForm* __restrict__ training_xforms,
	Matrix<float, 3, 4> camera_matrix,
	Vector2f focal_length,
	float depth_scale,
	Array4f* __restrict__ rgba,
	float* __restrict__ depth,
	Array3f* __restrict__ normals,
	NerfPayload* payloads,
	PitchedPtr<NerfCoordinate> network_input,
	PitchedPtr<NerfCoordinate> network_gradient,
	const tcnn::network_precision_t* __restrict__ network_output_old,
	const tcnn::network_precision_t* __restrict__ network_output,
	const SH9RGB* __restrict__ sh_boundary,
	const float* __restrict__ density_out_boundary,
	const float* __restrict__ density_residual_boundary,
	uint32_t padded_output_width,
	uint32_t n_steps,
	ERenderMode render_mode,
	const uint8_t* __restrict__ density_grid,
	ENerfActivation rgb_activation,
	ENerfActivation density_activation,
	int show_accel,
	float min_transmittance,
	const bool* __restrict__ empty_mask,
	bool poisson_target
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	NerfPayload& payload = payloads[i];

	if (!payload.alive) {
		return;
	}

	Array4f local_rgba = rgba[i];
	float local_depth = depth[i];
	Array3f local_normal = normals[i];
	Vector3f origin = payload.origin;
	Vector3f cam_fwd = camera_matrix.col(2);
	// Composite in the last n steps
	uint32_t actual_n_steps = payload.n_steps;
	uint32_t j = 0;

	for (; j < actual_n_steps; ++j) {
		tcnn::vector_t<tcnn::network_precision_t, 4> local_network_output;
		local_network_output[0] = network_output[i + j * n_elements + 0 * stride];
		local_network_output[1] = network_output[i + j * n_elements + 1 * stride];
		local_network_output[2] = network_output[i + j * n_elements + 2 * stride];
		local_network_output[3] = network_output[i + j * n_elements + 3 * stride];
		const NerfCoordinate* input = network_input(i + j * n_elements);
		const NerfCoordinate* gradient = network_gradient(i + j * n_elements);
		Vector3f warped_pos = input->pos.p;
		Vector3f pos = unwarp_position(warped_pos, aabb);

		float T = 1.f - local_rgba.w();
		float dt = unwarp_dt(input->dt);
		float alpha;
		if (empty_mask && empty_mask[i + j * n_elements])
		{
			alpha = 0.0f;
		}
		else
		{
			if (density_out_boundary[i + j * n_elements] > 1e-9f) {

				float sourceval = network_to_density(float(local_network_output[3]), density_activation);
				float targetval = network_to_density(float(network_output_old[i + j * n_elements + 3 * stride]), density_activation);
				float val;

				if (poisson_target)
					val = min(max(targetval, sourceval), sourceval + density_residual_boundary[i + j * n_elements]);
				else
					val = sourceval + density_residual_boundary[i + j * n_elements];
				
				alpha = 1.f - __expf(-(val) * dt);
			}
			else {
				alpha = 1.f - __expf(-network_to_density(float(local_network_output[3]), density_activation) * dt);
			}
		}

		if (show_accel >= 0) {
			alpha = 1.f;
		}
		float weight = alpha * T;

		Array3f rgb = network_to_rgb(local_network_output, rgb_activation);

		// If there is a poisson residual, perform rgb correction
		Array3f rgb_residual = Array3f::Zero();
		float w_N = 0.f;
		float w_R = 0.f;
		if (density_out_boundary[i + j * n_elements] > 1e-9f) {
			const float alpha_N = 1.f - __expf(-network_to_density(float(local_network_output[3]), density_activation) * dt);
			const float alpha_R = 1.f - __expf(-density_out_boundary[i + j * n_elements] * dt);
			w_N = alpha_N / (alpha_N + alpha_R);
			w_R = alpha_R / (alpha_N + alpha_R);
			rgb_residual = evaluate_sh9(sh_boundary[i + j * n_elements], unwarp_direction(input->dir.d));
		}

		if (glow_mode) { // random grid visualizations ftw!
#if 0
			if (0) {  // extremely startrek edition
				float glow_y = (pos.y() - (glow_y_cutoff - 0.5f)) * 2.f;
				if (glow_y>1.f) glow_y=max(0.f,21.f-glow_y*20.f);
				if (glow_y>0.f) {
					float line;
					line =max(0.f,cosf(pos.y()*2.f*3.141592653589793f * 16.f)-0.95f);
					line+=max(0.f,cosf(pos.x()*2.f*3.141592653589793f * 16.f)-0.95f);
					line+=max(0.f,cosf(pos.z()*2.f*3.141592653589793f * 16.f)-0.95f);
					line+=max(0.f,cosf(pos.y()*4.f*3.141592653589793f * 16.f)-0.975f);
					line+=max(0.f,cosf(pos.x()*4.f*3.141592653589793f * 16.f)-0.975f);
					line+=max(0.f,cosf(pos.z()*4.f*3.141592653589793f * 16.f)-0.975f);
					glow_y=glow_y*glow_y*0.5f + glow_y*line*25.f;
					rgb.y()+=glow_y;
					rgb.z()+=glow_y*0.5f;
					rgb.x()+=glow_y*0.25f;
				}
			}
#endif
			float glow = 0.f;

			bool green_grid = glow_mode & 1;
			bool green_cutline = glow_mode & 2;
			bool mask_to_alpha = glow_mode & 4;

			// less used?
			bool radial_mode = glow_mode & 8;
			bool grid_mode = glow_mode & 16; // makes object rgb go black!

			{
				float dist;
				if (radial_mode) {
					dist = (pos - camera_matrix.col(3)).norm();
					dist = min(dist, (4.5f - pos.y()) * 0.333f);
				} else {
					dist = pos.y();
				}

				if (grid_mode) {
					glow = 1.f / max(1.f, dist);
				} else {
					float y = glow_y_cutoff - dist; // - (ii*0.005f);
					float mask = 0.f;
					if (y > 0.f) {
						y *= 80.f;
						mask = min(1.f, y);
						//if (mask_mode) {
						//	rgb.x()=rgb.y()=rgb.z()=mask; // mask mode
						//} else
						{
							if (green_cutline) {
								glow += max(0.f, 1.f - abs(1.f -y)) * 4.f;
							}

							if (y>1.f) {
								y = 1.f - (y - 1.f) * 0.05f;
							}

							if (green_grid) {
								glow += max(0.f, y / max(1.f, dist));
							}
						}
					}
					if (mask_to_alpha) {
						weight *= mask;
					}
				}
			}

			if (glow > 0.f) {
				float line;
				line  = max(0.f,cosf(pos.y()*2.f*3.141592653589793f * 16.f)-0.975f);
				line += max(0.f,cosf(pos.x()*2.f*3.141592653589793f * 16.f)-0.975f);
				line += max(0.f,cosf(pos.z()*2.f*3.141592653589793f * 16.f)-0.975f);
				line += max(0.f,cosf(pos.y()*4.f*3.141592653589793f * 16.f)-0.975f);
				line += max(0.f,cosf(pos.x()*4.f*3.141592653589793f * 16.f)-0.975f);
				line += max(0.f,cosf(pos.z()*4.f*3.141592653589793f * 16.f)-0.975f);
				line += max(0.f,cosf(pos.y()*8.f*3.141592653589793f * 16.f)-0.975f);
				line += max(0.f,cosf(pos.x()*8.f*3.141592653589793f * 16.f)-0.975f);
				line += max(0.f,cosf(pos.z()*8.f*3.141592653589793f * 16.f)-0.975f);
				line += max(0.f,cosf(pos.y()*16.f*3.141592653589793f * 16.f)-0.975f);
				line += max(0.f,cosf(pos.x()*16.f*3.141592653589793f * 16.f)-0.975f);
				line += max(0.f,cosf(pos.z()*16.f*3.141592653589793f * 16.f)-0.975f);
				if (grid_mode) {
					glow = /*glow*glow*0.75f + */ glow*line*15.f;
					rgb.y() = glow;
					rgb.z() = glow*0.5f;
					rgb.x() = glow*0.25f;
				} else {
					glow=glow*glow*0.25f + glow*line*15.f;
					rgb.y() += glow;
					rgb.z() += glow*0.5f;
					rgb.x() += glow*0.25f;
				}
			}
		} // glow

		if (render_mode == ERenderMode::Normals) {
			// Network input contains the gradient of the network output w.r.t. input.
			// So to compute density gradients, we need to apply the chain rule.
			// The normal is then in the opposite direction of the density gradient (i.e. the direction of decreasing density)
			Vector3f normal = -network_to_density_derivative(float(local_network_output[3]), density_activation) * gradient->pos.p;
			rgb = normal.normalized().array();
		} else if (render_mode == ERenderMode::Positions) {
			if (show_accel>=0) {
				uint32_t mip = max(show_accel, mip_from_pos(pos));
				uint32_t res = NERF_GRIDSIZE() >> mip;
				int ix = pos.x()*(res);
				int iy = pos.y()*(res);
				int iz = pos.z()*(res);
				default_rng_t rng(ix+iy*232323+iz*727272);
				rgb.x() = 1.f-mip*(1.f/(NERF_CASCADES()-1));
				rgb.y() = rng.next_float();
				rgb.z() = rng.next_float();
			} else {
				rgb = (pos.array() - Array3f::Constant(0.5f)) / 2.0f + Array3f::Constant(0.5f);
			}
		} else if (render_mode == ERenderMode::EncodingVis) {
			rgb = warped_pos.array();
		} else if (render_mode == ERenderMode::Depth) {
			float z = cam_fwd.dot(pos-origin) * depth_scale;
			rgb = {z, z, z};
		} else if (render_mode == ERenderMode::Distance) {
			float z = (pos-origin).norm() * depth_scale;
			rgb = {z, z, z};
		} else if (render_mode == ERenderMode::Stepsize) {
			float warped_dt = warp_dt(dt);
			rgb = {warped_dt,warped_dt,warped_dt};
		} else if (render_mode == ERenderMode::AO) {
			rgb = Array3f::Constant(alpha);
		}
		if (density_out_boundary[i + j * n_elements] > 1e-9f) {
			local_rgba.head<3>() += weight * (w_N * rgb + w_R * rgb_residual); 
		} else {
			local_rgba.head<3>() += rgb * weight;
		}
		local_rgba.w() += weight;

		if (weight > payload.max_weight) {
			payload.max_weight = weight;
			local_depth = cam_fwd.dot(pos - camera_matrix.col(3));
		}

		if (local_rgba.w() > (1.0f - min_transmittance)) {
			local_rgba /= local_rgba.w();
			break;
		}
	}

	if (j < n_steps) {
		payload.alive = false;
		payload.n_steps = j + current_step;
	}

	//if (empty_mask != nullptr && empty_mask[i] && (local_rgba.x() != 0 || local_rgba.y() != 0 || local_rgba.z() != 0) && local_rgba.w() != 0)
	//{
	//	//printf("%f %f %f %f %f\n", local_rgba.x(), local_rgba.y(), local_rgba.z(), local_rgba.w(), local_depth);
	//	local_rgba.x() = 1.0f;
	//	local_rgba.z() = 0.0f;
	//	local_rgba.y() = 0.0f;
	//	local_rgba.w() = 1.0f;
	//}

	//if (empty_mask != nullptr && empty_mask[i])
	//{

	//}

	rgba[i] = local_rgba;
	depth[i] = local_depth;
	normals[i] = local_normal;
}

static constexpr float UNIFORM_SAMPLING_FRACTION = 0.5f;

inline __device__ Vector2f sample_cdf_2d(Vector2f sample, uint32_t img, const Vector2i& res, const float* __restrict__ cdf_x_cond_y, const float* __restrict__ cdf_y, float* __restrict__ pdf) {
	if (sample.x() < UNIFORM_SAMPLING_FRACTION) {
		sample.x() /= UNIFORM_SAMPLING_FRACTION;
		return sample;
	}

	sample.x() = (sample.x() - UNIFORM_SAMPLING_FRACTION) / (1.0f - UNIFORM_SAMPLING_FRACTION);

	cdf_y += img * res.y();

	// First select row according to cdf_y
	uint32_t y = binary_search(sample.y(), cdf_y, res.y());
	float prev = y > 0 ? cdf_y[y-1] : 0.0f;
	float pmf_y = cdf_y[y] - prev;
	sample.y() = (sample.y() - prev) / pmf_y;

	cdf_x_cond_y += img * res.y() * res.x() + y * res.x();

	// Then, select col according to x
	uint32_t x = binary_search(sample.x(), cdf_x_cond_y, res.x());
	prev = x > 0 ? cdf_x_cond_y[x-1] : 0.0f;
	float pmf_x = cdf_x_cond_y[x] - prev;
	sample.x() = (sample.x() - prev) / pmf_x;

	if (pdf) {
		*pdf = pmf_x * pmf_y * res.prod();
	}

	return {((float)x + sample.x()) / (float)res.x(), ((float)y + sample.y()) / (float)res.y()};
}

inline __device__ float pdf_2d(Vector2f sample, uint32_t img, const Vector2i& res, const float* __restrict__ cdf_x_cond_y, const float* __restrict__ cdf_y) {
	Vector2i p = (sample.cwiseProduct(res.cast<float>())).cast<int>().cwiseMax(0).cwiseMin(res - Vector2i::Ones());

	cdf_y += img * res.y();
	cdf_x_cond_y += img * res.y() * res.x() + p.y() * res.x();

	float pmf_y = cdf_y[p.y()];
	if (p.y() > 0) {
		pmf_y -= cdf_y[p.y()-1];
	}

	float pmf_x = cdf_x_cond_y[p.x()];
	if (p.x() > 0) {
		pmf_x -= cdf_x_cond_y[p.x()-1];
	}

	// Probability mass of picking the pixel
	float pmf = pmf_x * pmf_y;

	// To convert to probability density, divide by area of pixel
	return UNIFORM_SAMPLING_FRACTION + pmf * res.prod() * (1.0f - UNIFORM_SAMPLING_FRACTION);
}

inline __device__ Vector2f nerf_random_image_pos_training(default_rng_t& rng, const Vector2i& resolution, bool snap_to_pixel_centers, const float* __restrict__ cdf_x_cond_y, const float* __restrict__ cdf_y, const Vector2i& cdf_res, uint32_t img, float* __restrict__ pdf = nullptr) {
	Vector2f xy = random_val_2d(rng);

	if (cdf_x_cond_y) {
		xy = sample_cdf_2d(xy, img, cdf_res, cdf_x_cond_y, cdf_y, pdf);
	} else if (pdf) {
		*pdf = 1.0f;
	}

	if (snap_to_pixel_centers) {
		xy = (xy.cwiseProduct(resolution.cast<float>()).cast<int>().cwiseMax(0).cwiseMin(resolution - Vector2i::Ones()).cast<float>() + Vector2f::Constant(0.5f)).cwiseQuotient(resolution.cast<float>());
	}
	return xy;
}

inline __device__ uint32_t image_idx(uint32_t base_idx, uint32_t n_rays, uint32_t n_rays_total, uint32_t n_training_images, const float* __restrict__ cdf = nullptr, float* __restrict__ pdf = nullptr) {
	if (cdf) {
		float sample = ld_random_val(base_idx + n_rays_total, 0xdeadbeef);
		// float sample = random_val(base_idx + n_rays_total);
		uint32_t img = binary_search(sample, cdf, n_training_images);

		if (pdf) {
			float prev = img > 0 ? cdf[img-1] : 0.0f;
			*pdf = (cdf[img] - prev) * n_training_images;
		}

		return img;
	}

	// return ((base_idx + n_rays_total) * 56924617 + 96925573) % n_training_images;

	// Neighboring threads in the warp process the same image. Increases locality.
	if (pdf) {
		*pdf = 1.0f;
	}
	return (((base_idx + n_rays_total) * n_training_images) / n_rays) % n_training_images;
}

inline __device__ Vector2i image_pos(const Vector2f& pos, const Vector2i& resolution) {
	return pos.cwiseProduct(resolution.cast<float>()).cast<int>().cwiseMin(resolution - Vector2i::Constant(1)).cwiseMax(0);
}

inline __device__ uint64_t pixel_idx(const Vector2i& pos, const Vector2i& resolution, uint32_t img) {
	return pos.x() + pos.y() * resolution.x() + img * (uint64_t)resolution.x() * resolution.y();
}

inline __device__ uint64_t pixel_idx(const Vector2f& xy, const Vector2i& resolution, uint32_t img) {
	return pixel_idx(image_pos(xy, resolution), resolution, img);
}

__global__ void generate_training_samples_nerf(
	const uint32_t n_rays,
	BoundingBox aabb,
	const uint32_t max_samples,
	const uint32_t n_rays_total,
	default_rng_t rng,
	const Ray* __restrict__ rays_in,
	uint32_t* __restrict__ ray_counter,
	uint32_t* __restrict__ numsteps_counter,
	uint32_t* __restrict__ ray_indices_out,
	Ray* __restrict__ rays_out,
	uint32_t* __restrict__ numsteps_out,
	PitchedPtr<NerfCoordinate> coords_out,
	Vector2i resolution,
	const uint32_t n_training_images,
	const TrainingImageMetadata* __restrict__ metadata,
	const TrainingXForm* training_xforms,
	const uint8_t* __restrict__ density_grid,
	bool max_level_rand_training,
	float* __restrict__ max_level_ptr,
	bool snap_to_pixel_centers,
	bool train_envmap,
	float cone_angle_constant,
	const float* __restrict__ distortion_data,
	const Vector2i distortion_resolution,
	const float* __restrict__ cdf_x_cond_y,
	const float* __restrict__ cdf_y,
	const float* __restrict__ cdf_img,
	const Vector2i cdf_res,
	const __half* __restrict__ training_images
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_rays) return;

	uint32_t img = image_idx(i, n_rays, n_rays_total, n_training_images, cdf_img);

	rng.advance(i * N_MAX_RANDOM_SAMPLES_PER_RAY());
	Vector2f xy = nerf_random_image_pos_training(rng, resolution, snap_to_pixel_centers, cdf_x_cond_y, cdf_y, cdf_res, img);

	// Negative values indicate masked-away regions
	if ((float)training_images[pixel_idx(xy, resolution, img)*4] < 0.0f) {
		return;
	}

	float max_level = max_level_rand_training ? (random_val(rng) * 2.0f) : 1.0f; // Multiply by 2 to ensure 50% of training is at max level

	float motionblur_time = random_val(rng);

	const Vector2f focal_length = metadata[img].focal_length;
	const Vector2f principal_point = metadata[img].principal_point;
	const Vector3f light_dir_warped = warp_direction(metadata[img].light_dir);
	const CameraDistortion camera_distortion = metadata[img].camera_distortion;

	const Matrix<float, 3, 4> xform = get_xform_given_rolling_shutter(training_xforms[img], metadata[img].rolling_shutter, xy, motionblur_time);

	Ray ray;
	if (rays_in) {
		// Rays have been explicitly supplied. Read them.
		ray = rays_in[pixel_idx(xy, resolution, img)];

		/* DEBUG - compare the stored rays to the computed ones
		const Matrix<float, 3, 4> xform = get_xform_given_rolling_shutter(training_xforms[img], metadata[img].rolling_shutter, xy, 0.f);
		Ray ray2;
		ray2.o = xform.col(3);
		ray2.d = f_theta_distortion(xy, principal_point, camera_distortion);
		ray2.d = (xform.block<3, 3>(0, 0) * ray2.d).normalized();
		if (i==1000) {
			printf("\n%d uv %0.3f,%0.3f pixel %0.2f,%0.2f transform from [%0.5f %0.5f %0.5f] to [%0.5f %0.5f %0.5f]\n"
				" origin    [%0.5f %0.5f %0.5f] vs [%0.5f %0.5f %0.5f]\n"
				" direction [%0.5f %0.5f %0.5f] vs [%0.5f %0.5f %0.5f]\n"
			, img,xy.x(), xy.y(), xy.x()*resolution.x(), xy.y()*resolution.y(),
				training_xforms[img].start.col(3).x(),training_xforms[img].start.col(3).y(),training_xforms[img].start.col(3).z(),
				training_xforms[img].end.col(3).x(),training_xforms[img].end.col(3).y(),training_xforms[img].end.col(3).z(),
				ray.o.x(),ray.o.y(),ray.o.z(),
				ray2.o.x(),ray2.o.y(),ray2.o.z(),
				ray.d.x(),ray.d.y(),ray.d.z(),
				ray2.d.x(),ray2.d.y(),ray2.d.z());
		}
		*/
	} else {
		// Rays need to be inferred from the camera matrix
		ray.o = xform.col(3);
		if (camera_distortion.mode==ECameraDistortionMode::FTheta) {
			ray.d = f_theta_undistortion(xy, principal_point, camera_distortion, {0.f, 0.f, 1.f});
		} else {
			ray.d = {
				(xy.x()-principal_point.x())*resolution.x() / focal_length.x(),
				(xy.y()-principal_point.y())*resolution.y() / focal_length.y(),
				1.0f,
			};
			if (camera_distortion.mode==ECameraDistortionMode::Iterative) {
				iterative_camera_undistortion(camera_distortion.params, &ray.d.x(), &ray.d.y());
			}
		}
		if (distortion_data) {
			ray.d.head<2>() += read_image<2>(distortion_data, distortion_resolution, xy);
		}

		ray.d = (xform.block<3, 3>(0, 0) * ray.d).normalized();
	}

	Vector2f tminmax = aabb.ray_intersect(ray.o, ray.d);
	float cone_angle = calc_cone_angle(ray.d.dot(xform.col(2)), focal_length, cone_angle_constant);

	// The near distance prevents learning of camera-specific fudge right in front of the camera
	tminmax.x() = fmaxf(tminmax.x(), 0.0f);

	float startt = tminmax.x();
	startt += calc_dt(startt, cone_angle) * random_val(rng);
	Vector3f idir = ray.d.cwiseInverse();

	// first pass to compute an accurate number of steps
	uint32_t j = 0;
	float t=startt;
	Vector3f pos;

	while (aabb.contains(pos = ray.o + t * ray.d) && j < NERF_STEPS()) {
		float dt = calc_dt(t, cone_angle);
		uint32_t mip = mip_from_dt(dt, pos);
		if (density_grid_occupied_at(pos, density_grid, mip)) {
			++j;
			t += dt;
		} else {
			uint32_t res = NERF_GRIDSIZE()>>mip;
			t = advance_to_next_voxel(t, cone_angle, pos, ray.d, idir, res);
		}
	}
	if (j == 0 && !train_envmap) {
		return;
	}
	uint32_t numsteps = j;
	uint32_t base = atomicAdd(numsteps_counter, numsteps);	 // first entry in the array is a counter
	if (base + numsteps > max_samples) {
		return;
	}

	coords_out += base;

	uint32_t ray_idx = atomicAdd(ray_counter, 1);

	ray_indices_out[ray_idx] = i;
	rays_out[ray_idx] = ray;
	numsteps_out[ray_idx*2+0] = numsteps;
	numsteps_out[ray_idx*2+1] = base;

	Vector3f warped_dir = warp_direction(ray.d);
	t=startt;
	j=0;
	while (aabb.contains(pos = ray.o + t * ray.d) && j < numsteps) {
		float dt = calc_dt(t, cone_angle);
		uint32_t mip = mip_from_dt(dt, pos);
		if (density_grid_occupied_at(pos, density_grid, mip)) {
			coords_out(j)->set_with_optional_light_dir(warp_position(pos, aabb), warped_dir, warp_dt(dt), light_dir_warped, coords_out.stride_in_bytes);
			++j;
			t += dt;
		} else {
			uint32_t res = NERF_GRIDSIZE()>>mip;
			t = advance_to_next_voxel(t, cone_angle, pos, ray.d, idir, res);
		}
	}
	if (max_level_rand_training) {
		max_level_ptr += base;
		for (j = 0; j < numsteps; ++j) {
			max_level_ptr[j] = max_level;
		}
	}
}

__global__ void generate_training_samples_nerf_distill(
	const uint32_t n_rays,
	BoundingBox aabb,
	const uint32_t max_samples,
	const uint32_t n_rays_total,
	default_rng_t rng,
	const Ray* __restrict__ rays_in,
	uint32_t* __restrict__ ray_counter,
	uint32_t* __restrict__ numsteps_counter,
	uint32_t* __restrict__ ray_indices_out,
	Ray* __restrict__ rays_out,
	uint32_t* __restrict__ numsteps_out,
	PitchedPtr<NerfCoordinate> coords_out,
	Vector2i resolution,
	const uint32_t n_training_images,
	const TrainingImageMetadata* __restrict__ metadata,
	const TrainingXForm* training_xforms,
	const uint8_t* __restrict__ density_grid,
	bool max_level_rand_training,
	float* __restrict__ max_level_ptr,
	bool snap_to_pixel_centers,
	bool train_envmap,
	float cone_angle_constant,
	const float* __restrict__ distortion_data,
	const Vector2i distortion_resolution,
	const float* __restrict__ cdf_x_cond_y,
	const float* __restrict__ cdf_y,
	const float* __restrict__ cdf_img,
	const Vector2i cdf_res,
	const __half* __restrict__ training_images,
	Distiller** cage_infos,
	int num_cages,
	int* to_duplicate,
	AuxRay* auxrays,
	int allowed_aux_rays,
	AuxSample* auxsamples
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_rays) return;

	uint32_t img = image_idx(i, n_rays, n_rays_total, n_training_images, cdf_img);

	rng.advance(i * N_MAX_RANDOM_SAMPLES_PER_RAY());
	Vector2f xy = nerf_random_image_pos_training(rng, resolution, snap_to_pixel_centers, cdf_x_cond_y, cdf_y, cdf_res, img);

	// Negative values indicate masked-away regions
	if ((float)training_images[pixel_idx(xy, resolution, img) * 4] < 0.0f) {
		return;
	}

	float max_level = max_level_rand_training ? (random_val(rng) * 2.0f) : 1.0f; // Multiply by 2 to ensure 50% of training is at max level

	float motionblur_time = random_val(rng);

	const Vector2f focal_length = metadata[img].focal_length;
	const Vector2f principal_point = metadata[img].principal_point;
	const Vector3f light_dir_warped = warp_direction(metadata[img].light_dir);
	const CameraDistortion camera_distortion = metadata[img].camera_distortion;

	const Matrix<float, 3, 4> xform = get_xform_given_rolling_shutter(training_xforms[img], metadata[img].rolling_shutter, xy, motionblur_time);

	Ray ray;
	if (rays_in) {
		// Rays have been explicitly supplied. Read them.
		ray = rays_in[pixel_idx(xy, resolution, img)];
	}
	else {
		// Rays need to be inferred from the camera matrix
		ray.o = xform.col(3);
		if (camera_distortion.mode == ECameraDistortionMode::FTheta) {
			ray.d = f_theta_undistortion(xy, principal_point, camera_distortion, { 0.f, 0.f, 1.f });
		}
		else {
			ray.d = {
				(xy.x() - principal_point.x()) * resolution.x() / focal_length.x(),
				(xy.y() - principal_point.y()) * resolution.y() / focal_length.y(),
				1.0f,
			};
			if (camera_distortion.mode == ECameraDistortionMode::Iterative) {
				iterative_camera_undistortion(camera_distortion.params, &ray.d.x(), &ray.d.y());
			}
		}
		if (distortion_data) {
			ray.d.head<2>() += read_image<2>(distortion_data, distortion_resolution, xy);
		}

		ray.d = (xform.block<3, 3>(0, 0) * ray.d).normalized();
	}

	Vector2f tminmax = aabb.ray_intersect(ray.o, ray.d);
	float cone_angle = calc_cone_angle(ray.d.dot(xform.col(2)), focal_length, cone_angle_constant);

	// The near distance prevents learning of camera-specific fudge right in front of the camera
	tminmax.x() = fmaxf(tminmax.x(), 0.0f);

	float startt = tminmax.x();
	startt += calc_dt(startt, cone_angle) * random_val(rng);
	Vector3f idir = ray.d.cwiseInverse();

	// first pass to compute an accurate number of steps
	uint32_t j = 0;
	float t = startt;
	Vector3f pos;

	uint32_t unique_combos[20];
	bool is_emptying[20];
	uint32_t unique_length = 0;

	while (aabb.contains(pos = ray.o + t * ray.d) && j < NERF_STEPS()) {
		float dt = calc_dt(t, cone_angle);
		uint32_t mip = mip_from_dt(dt, pos);
		if (density_grid_occupied_at(pos, density_grid, mip)) {
			++j;
			t += dt;
		}
		else {
			uint32_t res = NERF_GRIDSIZE() >> mip;
			t = advance_to_next_voxel(t, cone_angle, pos, ray.d, idir, res);
		}

		recListIgnored(false, is_emptying, unique_combos, unique_length, pos, cage_infos, 0, num_cages, 0);
	}

	if (unique_length > 0)
	{
		AuxRay auxray;
		auxray.ray = ray;
		auxray.cone_angle = cone_angle;
		auxray.startt = startt;
		auxray.max_level = max_level;
		auxray.light_dir_warped = light_dir_warped;
		auxray.max_level_rand_training = max_level_rand_training;
		auxray.ray_index = i;

		int added_rays = unique_length + 1;
		int off = atomicAdd(to_duplicate, added_rays);

		for (int i = 0; (off + i) < min(off + added_rays, allowed_aux_rays); i++)
		{
			if (i == added_rays - 1)
			{
				auxray.ignoring = 0;
				auxray.emptying = false;
			}
			else
			{
				auxray.ignoring = unique_combos[i];
				auxray.emptying = is_emptying[i];
			}
			auxrays[off + i] = auxray;
		}
		return;
	}

	if (j == 0 && !train_envmap) {
		return;
	}

	uint32_t numsteps = j;
	uint32_t base = atomicAdd(numsteps_counter, numsteps);	 // first entry in the array is a counter
	if (base + numsteps > max_samples) {
		return;
	}

	coords_out += base;
	auxsamples += base;

	uint32_t ray_idx = atomicAdd(ray_counter, 1);

	ray_indices_out[ray_idx] = i;
	rays_out[ray_idx] = ray;
	numsteps_out[ray_idx * 2 + 0] = numsteps;
	numsteps_out[ray_idx * 2 + 1] = base;

	Vector3f warped_dir = warp_direction(ray.d);
	t = startt;
	j = 0;
	while (aabb.contains(pos = ray.o + t * ray.d) && j < numsteps) {
		float dt = calc_dt(t, cone_angle);
		uint32_t mip = mip_from_dt(dt, pos);
		if (density_grid_occupied_at(pos, density_grid, mip)) {
			coords_out(j)->set_with_optional_light_dir(warp_position(pos, aabb), warped_dir, warp_dt(dt), light_dir_warped, coords_out.stride_in_bytes);

			for (int c = 0; c < num_cages; c++)
			{
				if (cage_infos[c]->in_target(pos, false)) // Mapping is broken, stop looking
				{
					auxsamples[j].overwritten = true;
					break;
				}
			}

			++j;
			t += dt;
		}
		else {
			uint32_t res = NERF_GRIDSIZE() >> mip;
			t = advance_to_next_voxel(t, cone_angle, pos, ray.d, idir, res);
		}
	}
	if (max_level_rand_training) {
		max_level_ptr += base;
		for (j = 0; j < numsteps; ++j) {
			max_level_ptr[j] = max_level;
		}
	}
}

__global__ void generate_extra_training_samples_nerf_distill(
	const uint32_t n_rays,
	BoundingBox aabb,
	const uint32_t max_samples,
	uint32_t* __restrict__ ray_counter,
	uint32_t* __restrict__ numsteps_counter,
	uint32_t* __restrict__ ray_indices_out,
	Ray* __restrict__ rays_out,
	uint32_t* __restrict__ numsteps_out,
	PitchedPtr<NerfCoordinate> coords_out,
	const uint8_t* __restrict__ density_grid,
	bool max_level_rand_training,
	float* __restrict__ max_level_ptr,
	bool train_envmap,
	Distiller** cage_infos,
	int num_cages,
	AuxRay* auxrays,
	AuxSample* auxsamples
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_rays) return;

	AuxRay auxray = auxrays[i];


	float max_level = auxray.max_level;
	const Vector3f light_dir_warped = auxray.light_dir_warped;
	Ray ray = auxray.ray;

	float cone_angle = auxray.cone_angle;

	float startt = auxray.startt;
	Vector3f idir = ray.d.cwiseInverse();

	// first pass to compute an accurate number of steps
	uint32_t j = 0;
	float t = startt;
	Vector3f pos;

	uint32_t ignored = auxray.ignoring;

	while (aabb.contains(pos = ray.o + t * ray.d) && j < NERF_STEPS()) {
		float dt = calc_dt(t, cone_angle);

		auto transPos = pos;
		for (int c = 0; c < num_cages; c++)
		{
			if (cage_infos[c]->in_source(transPos, false))
			{
				if ((ignored & (1 << c)) == 0) // do apply this transform
					transPos = cage_infos[c]->map(transPos, false);
				else if (cage_infos[c]->in_target(transPos, false))
				{
					transPos = pos;
					break;
				}
			}
			else if (cage_infos[c]->in_target(transPos, false))
			{
				transPos = pos;
				break;
			}
		}

		bool valid = aabb.contains(warp_position(transPos, aabb));

		uint32_t mip = mip_from_dt(dt, transPos);
		if (valid && density_grid_occupied_at(transPos, density_grid, mip)) {
			++j;
			t += dt;
		}
		else {
			uint32_t res = NERF_GRIDSIZE() >> mip_from_dt(dt, pos);
			t = advance_to_next_voxel(t, cone_angle, pos, ray.d, idir, res);
		}
	}

	if (j == 0 && !train_envmap) {
		return;
	}
	uint32_t numsteps = j;
	uint32_t base = atomicAdd(numsteps_counter, numsteps);	 // first entry in the array is a counter
	if (base + numsteps > max_samples) {
		return;
	}

	coords_out += base;
	auxsamples += base;

	uint32_t ray_idx = atomicAdd(ray_counter, 1);

	ray_indices_out[ray_idx] = auxray.ray_index;
	rays_out[ray_idx] = ray;
	numsteps_out[ray_idx * 2 + 0] = numsteps;
	numsteps_out[ray_idx * 2 + 1] = base;

	Vector3f warped_dir = warp_direction(ray.d);
	t = startt;
	j = 0;
	while (aabb.contains(pos = ray.o + t * ray.d) && j < numsteps) {
		float dt = calc_dt(t, cone_angle);

		bool overwritten = false;
		auto transPos = pos;
		uint32_t inside = 0;
		for (int c = 0; c < num_cages; c++)
		{
			if (cage_infos[c]->in_source(transPos, false))
			{
				inside |= (1 << c);

				if ((ignored & (1 << c)) == 0) // do apply this transform
					transPos = cage_infos[c]->map(transPos, false);
				else if (cage_infos[c]->in_target(transPos, false))
				{
					transPos = pos;
					overwritten = true;
					break;
				}
			}
			else if (cage_infos[c]->in_target(transPos, false)) // Mapping is broken, stop looking
			{
				transPos = pos;
				overwritten = true;
				break;
			}
		}

		bool valid = aabb.contains(warp_position(transPos, aabb));

		uint32_t mip = mip_from_dt(dt, transPos);
		if (valid && density_grid_occupied_at(transPos, density_grid, mip)) {
			coords_out(j)->set_with_optional_light_dir(warp_position(transPos, aabb), warped_dir, warp_dt(dt), light_dir_warped, coords_out.stride_in_bytes);

			if (ignored != 0)
			{
				// TODO: store per ray if we are emptying or not.
				// We empty if the ignored destination space is a non-copying cage.
				auxsamples[j].emptying = auxray.emptying;

				if ((inside & ignored) == ignored)
					auxsamples[j].inside = true;
				else // didn't make it to dedicated space
					auxsamples[j].outside = true;
			}
			auxsamples[j].overwritten = overwritten; // mapping is impossible, color from fallback

			++j;
			t += dt;
		}
		else {
			uint32_t res = NERF_GRIDSIZE() >> mip_from_dt(dt, pos);
			t = advance_to_next_voxel(t, cone_angle, pos, ray.d, idir, res);
		}
	}
	if (max_level_rand_training) {
		max_level_ptr += base;
		for (j = 0; j < numsteps; ++j) {
			max_level_ptr[j] = max_level;
		}
	}
}

__device__ LossAndGradient loss_and_gradient(const Vector3f& target, const Vector3f& prediction, ELossType loss_type) {
	switch (loss_type) {
		case ELossType::RelativeL2:  return relative_l2_loss(target, prediction); break;
		case ELossType::L1:          return l1_loss(target, prediction); break;
		case ELossType::Mape:        return mape_loss(target, prediction); break;
		case ELossType::Smape:       return smape_loss(target, prediction); break;
		// Note: we divide the huber loss by a factor of 5 such that its L2 region near zero
		// matches with the L2 loss and error numbers become more comparable. This allows reading
		// off dB numbers of ~converged models and treating them as approximate PSNR to compare
		// with other NeRF methods. Self-normalizing optimizers such as Adam are agnostic to such
		// constant factors; optimization is therefore unaffected.
		case ELossType::Huber:       return huber_loss(target, prediction, 0.1f) / 5.0f; break;
		case ELossType::LogL1:       return log_l1_loss(target, prediction); break;
		default: case ELossType::L2: return l2_loss(target, prediction); break;
	}
}

inline __device__ Array3f composit_and_lerp(Vector2f pos, const Vector2i& resolution, uint32_t img, const __half* training_images, const Array3f& background_color, const Array3f& exposure_scale = Array3f::Ones()) {
	pos = (pos.cwiseProduct(resolution.cast<float>()) - Vector2f::Constant(0.5f)).cwiseMax(0.0f).cwiseMin(resolution.cast<float>() - Vector2f::Constant(1.0f + 1e-4f));

	const Vector2i pos_int = pos.cast<int>();
	const Vector2f weight = pos - pos_int.cast<float>();

	const Vector2i idx = pos_int.cwiseMin(resolution - Vector2i::Constant(2)).cwiseMax(0);

	auto read_val = [&](const Vector2i& p) {
		__half val[4];
		*(uint64_t*)&val[0] = ((uint64_t*)training_images)[pixel_idx(p, resolution, img)];
		return Array3f{val[0], val[1], val[2]} * exposure_scale + background_color * (1.0f - (float)val[3]);
	};

	return (
		(1 - weight.x()) * (1 - weight.y()) * read_val({idx.x(), idx.y()}) +
		(weight.x()) * (1 - weight.y()) * read_val({idx.x()+1, idx.y()}) +
		(1 - weight.x()) * (weight.y()) * read_val({idx.x(), idx.y()+1}) +
		(weight.x()) * (weight.y()) * read_val({idx.x()+1, idx.y()+1})
	);
}

inline __device__ Array3f composit(Vector2f pos, const Vector2i& resolution, uint32_t img, const __half* training_images, const Array3f& background_color, const Array3f& exposure_scale = Array3f::Ones()) {
	auto read_val = [&](const Vector2i& p) {
		__half val[4];
		*(uint64_t*)&val[0] = ((uint64_t*)training_images)[pixel_idx(p, resolution, img)];
		return Array3f{val[0], val[1], val[2]} * exposure_scale + background_color * (1.0f - (float)val[3]);
	};

	return read_val(image_pos(pos, resolution));
}

inline __device__ Array4f read_rgba(Vector2f pos, const Vector2i& resolution, uint32_t img, const __half* training_images) {
	auto read_val = [&](const Vector2i& p) {
		__half val[4];
		*(uint64_t*)&val[0] = ((uint64_t*)training_images)[pixel_idx(p, resolution, img)];
		return Array4f{val[0], val[1], val[2], val[3]};
	};

	return read_val(image_pos(pos, resolution));
}

__global__ void compute_loss_kernel_train_nerf(
	const uint32_t n_rays,
	BoundingBox aabb,
	const uint32_t n_rays_total,
	default_rng_t rng,
	const uint32_t max_samples_compacted,
	const uint32_t* __restrict__ rays_counter,
	float loss_scale,
	int padded_output_width,
	const float* __restrict__ envmap_data,
	float* __restrict__ envmap_gradient,
	const Vector2i envmap_resolution,
	ELossType envmap_loss_type,
	Array3f background_color,
	EColorSpace color_space,
	bool train_with_random_bg_color,
	bool train_in_linear_colors,
	const __half* __restrict__ training_images,
	const uint32_t n_training_images,
	Vector2i resolution,
	const tcnn::network_precision_t* network_output,
	uint32_t* __restrict__ numsteps_counter,
	const uint32_t* __restrict__ ray_indices_in,
	const Ray* __restrict__ rays_in,
	uint32_t* __restrict__ numsteps_in,
	PitchedPtr<const NerfCoordinate> coords_in,
	PitchedPtr<NerfCoordinate> coords_out,
	tcnn::network_precision_t* dloss_doutput,
	ELossType loss_type,
	float* __restrict__ loss_output,
	bool max_level_rand_training,
	float* __restrict__ max_level_compacted_ptr,
	ENerfActivation rgb_activation,
	ENerfActivation density_activation,
	bool snap_to_pixel_centers,
	float* __restrict__ error_map,
	const float* __restrict__ cdf_x_cond_y,
	const float* __restrict__ cdf_y,
	const float* __restrict__ cdf_img,
	const Vector2i error_map_res,
	const Vector2i error_map_cdf_res,
	const float* __restrict__ sharpness_data,
	Eigen::Vector2i sharpness_resolution,
	float* __restrict__ sharpness_grid,
	float* __restrict__ density_grid,
	const float* __restrict__ mean_density_ptr,
	const Eigen::Array3f* __restrict__ exposure,
	Eigen::Array3f* __restrict__ exposure_gradient,
	const float near_distance
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= *rays_counter) { return; }

	// grab the number of samples for this ray, and the first sample
	uint32_t numsteps = numsteps_in[i*2+0];
	uint32_t base = numsteps_in[i*2+1];

	coords_in += base;
	network_output += base * padded_output_width;

	float T = 1.f;

	float EPSILON = 1e-4f;

	Array3f rgb_ray = Array3f::Zero();
	Vector3f hitpoint = Vector3f::Zero();

	uint32_t compacted_numsteps = 0;
	for (; compacted_numsteps < numsteps; ++compacted_numsteps) {
		if (T < EPSILON) {
			break;
		}

		const tcnn::vector_t<tcnn::network_precision_t, 4> local_network_output = *(tcnn::vector_t<tcnn::network_precision_t, 4>*)network_output;
		const Array3f rgb = network_to_rgb(local_network_output, rgb_activation);
		const Vector3f pos = unwarp_position(coords_in.ptr->pos.p, aabb);
		const float dt = unwarp_dt(coords_in.ptr->dt);

		float density = network_to_density(float(local_network_output[3]), density_activation);


		const float alpha = 1.f - __expf(-density * dt);
		const float weight = alpha * T;
		rgb_ray += weight * rgb;
		hitpoint += weight * pos;
		T *= (1.f - alpha);

		network_output += padded_output_width;
		coords_in += 1;
	}
	hitpoint /= (1.0f - T);

	// Must be same seed as above to obtain the same
	// background color.
	uint32_t ray_idx = ray_indices_in[i];
	rng.advance(ray_idx * N_MAX_RANDOM_SAMPLES_PER_RAY());

	float img_pdf = 1.0f;
	uint32_t img = image_idx(ray_idx, n_rays, n_rays_total, n_training_images, cdf_img, &img_pdf);

	float xy_pdf = 1.0f;
	Vector2f xy = nerf_random_image_pos_training(rng, resolution, snap_to_pixel_centers, cdf_x_cond_y, cdf_y, error_map_cdf_res, img, &xy_pdf);
	float max_level = max_level_rand_training ? (random_val(rng) * 2.0f) : 1.0f; // Multiply by 2 to ensure 50% of training is at max level

	if (train_with_random_bg_color) {
		background_color = random_val_3d(rng);
	}
	Array3f pre_envmap_background_color = background_color = srgb_to_linear(background_color);

	// Composit background behind envmap
	Array4f envmap_value;
	Vector3f dir;
	if (envmap_data) {
		dir = rays_in[i].d;
		envmap_value = read_envmap(envmap_data, envmap_resolution, dir);
		background_color = envmap_value.head<3>() + background_color * (1.0f - envmap_value.w());
	}

	Array3f exposure_scale = (0.6931471805599453f * exposure[img]).exp();
	// Array3f rgbtarget = composit_and_lerp(xy, resolution, img, training_images, background_color, exposure_scale);
	// Array3f rgbtarget = composit(xy, resolution, img, training_images, background_color, exposure_scale);
	Array4f texsamp = read_rgba(xy, resolution, img, training_images);

	Array3f rgbtarget;
	if (train_in_linear_colors || color_space == EColorSpace::Linear) {
		rgbtarget = exposure_scale * texsamp.head<3>() + (1.0f - texsamp.w()) * background_color;

		if (!train_in_linear_colors) {
			rgbtarget = linear_to_srgb(rgbtarget);
			background_color = linear_to_srgb(background_color);
		}
	} else if (color_space == EColorSpace::SRGB) {
		background_color = linear_to_srgb(background_color);
		if (texsamp.w() > 0) {
			rgbtarget = linear_to_srgb(exposure_scale * texsamp.head<3>() / texsamp.w()) * texsamp.w() + (1.0f - texsamp.w()) * background_color;
		} else {
			rgbtarget = background_color;
		}
	}

	if (compacted_numsteps == numsteps) {
		// support arbitrary background colors
		rgb_ray += T * background_color;
	}

	// Step again, this time computing loss
	network_output -= padded_output_width * compacted_numsteps; // rewind the pointer
	coords_in -= compacted_numsteps;

	uint32_t compacted_base = atomicAdd(numsteps_counter, compacted_numsteps); // first entry in the array is a counter
	compacted_numsteps = min(max_samples_compacted - min(max_samples_compacted, compacted_base), compacted_numsteps);
	numsteps_in[i*2+0] = compacted_numsteps;
	numsteps_in[i*2+1] = compacted_base;
	if (compacted_numsteps == 0) {
		return;
	}

	max_level_compacted_ptr += compacted_base;
	coords_out += compacted_base;

	dloss_doutput += compacted_base * padded_output_width;

	LossAndGradient lg = loss_and_gradient(rgbtarget, rgb_ray, loss_type);
	lg.loss /= img_pdf * xy_pdf;

	// Note: dividing the gradient by the PDF would cause unbiased loss estimates.
	// Essentially: variance reduction, but otherwise the same optimization.
	// We _dont_ want that. If importance sampling is enabled, we _do_ actually want
	// to change the weighting of the loss function. So don't divide.
	// lg.gradient /= img_pdf * xy_pdf;

	float mean_loss = lg.loss.mean();
	if (loss_output) {
		loss_output[i] = mean_loss / (float)n_rays;
	}

	if (error_map) {
		const Vector2f pos = (xy.cwiseProduct(error_map_res.cast<float>()) - Vector2f::Constant(0.5f)).cwiseMax(0.0f).cwiseMin(error_map_res.cast<float>() - Vector2f::Constant(1.0f + 1e-4f));
		const Vector2i pos_int = pos.cast<int>();
		const Vector2f weight = pos - pos_int.cast<float>();

		Vector2i idx = pos_int.cwiseMin(resolution - Vector2i::Constant(2)).cwiseMax(0);

		auto deposit_val = [&](int x, int y, float val) {
			atomicAdd(&error_map[img * error_map_res.prod() + y * error_map_res.x() + x], val);
		};

		if (sharpness_data && aabb.contains(hitpoint)) {
			Vector2i sharpness_pos = xy.cwiseProduct(sharpness_resolution.cast<float>()).cast<int>().cwiseMax(0).cwiseMin(sharpness_resolution - Vector2i::Constant(1));
			float sharp = sharpness_data[img * sharpness_resolution.prod() + sharpness_pos.y() * sharpness_resolution.x() + sharpness_pos.x()] + 1e-6f;

			// The maximum value of positive floats interpreted in uint format is the same as the maximum value of the floats.
			float grid_sharp = __uint_as_float(atomicMax((uint32_t*)&cascaded_grid_at(hitpoint, sharpness_grid, mip_from_pos(hitpoint)), __float_as_uint(sharp)));
			grid_sharp = fmaxf(sharp, grid_sharp); // atomicMax returns the old value, so compute the new one locally.

			mean_loss *= fmaxf(sharp / grid_sharp, 0.01f);
		}

		deposit_val(idx.x(),   idx.y(),   (1 - weight.x()) * (1 - weight.y()) * mean_loss);
		deposit_val(idx.x()+1, idx.y(),        weight.x()  * (1 - weight.y()) * mean_loss);
		deposit_val(idx.x(),   idx.y()+1, (1 - weight.x()) *      weight.y()  * mean_loss);
		deposit_val(idx.x()+1, idx.y()+1,      weight.x()  *      weight.y()  * mean_loss);
	}

	loss_scale /= n_rays;

	const float output_l2_reg = rgb_activation == ENerfActivation::Exponential ? 1e-4f : 0.0f;
	const float output_l1_reg_density = *mean_density_ptr < NERF_MIN_OPTICAL_THICKNESS() ? 1e-4f : 0.0f;

	// now do it again computing gradients
	Array3f rgb_ray2 = { 0.f,0.f,0.f };
	T = 1.f;
	Vector3f origin = rays_in[i].o;
	for (uint32_t j = 0; j < compacted_numsteps; ++j) {
		if (max_level_rand_training) {
			max_level_compacted_ptr[j] = max_level;
		}
		// Compact network inputs
		NerfCoordinate* coord_out = coords_out(j);
		const NerfCoordinate* coord_in = coords_in(j);
		coord_out->copy_with_optional_light_dir(*coord_in, coords_out.stride_in_bytes);
		float dt = unwarp_dt(coord_in->dt);
		const tcnn::vector_t<tcnn::network_precision_t, 4> local_network_output = *(tcnn::vector_t<tcnn::network_precision_t, 4>*)network_output;
		const Array3f rgb = network_to_rgb(local_network_output, rgb_activation);
		const float density = network_to_density(float(local_network_output[3]), density_activation);
		const float alpha = 1.f - __expf(-density * dt);
		const float weight = alpha * T;
		rgb_ray2 += weight * rgb;
		T *= (1.f - alpha);

		// we know the suffix of this ray compared to where we are up to. note the suffix depends on this step's alpha as suffix = (1-alpha)*(somecolor), so dsuffix/dalpha = -somecolor = -suffix/(1-alpha)
		const Array3f suffix = rgb_ray - rgb_ray2;
		const Array3f dloss_by_drgb = weight * lg.gradient;

		tcnn::vector_t<tcnn::network_precision_t, 4> local_dL_doutput;

		// chain rule to go from dloss/drgb to dloss/dmlp_output
		local_dL_doutput[0] = loss_scale * (dloss_by_drgb.x() * network_to_rgb_derivative(local_network_output[0], rgb_activation) + fmaxf(0.0f, output_l2_reg * (float)local_network_output[0])); // Penalize way too large color values
		local_dL_doutput[1] = loss_scale * (dloss_by_drgb.y() * network_to_rgb_derivative(local_network_output[1], rgb_activation) + fmaxf(0.0f, output_l2_reg * (float)local_network_output[1]));
		local_dL_doutput[2] = loss_scale * (dloss_by_drgb.z() * network_to_rgb_derivative(local_network_output[2], rgb_activation) + fmaxf(0.0f, output_l2_reg * (float)local_network_output[2]));

		float density_derivative = network_to_density_derivative(float(local_network_output[3]), density_activation);
		float dloss_by_dmlp = density_derivative * (
			dt * lg.gradient.matrix().dot((T * rgb - suffix).matrix())
		);

		//static constexpr float mask_supervision_strength = 1.f; // we are already 'leaking' mask information into the nerf via the random bg colors; setting this to eg between 1 and  100 encourages density towards 0 in such regions.
		//dloss_by_dmlp += (texsamp.w()<0.001f) ? mask_supervision_strength * weight : 0.f ;

		local_dL_doutput[3] =
			loss_scale * dloss_by_dmlp +
			(float(local_network_output[3]) < 0.0f ? -output_l1_reg_density : 0.0f) +
			(float(local_network_output[3]) > -10.0f && (unwarp_position(coord_in->pos.p, aabb) - origin).norm() < near_distance ? 1e-4f : 0.0f);
			;

		*(tcnn::vector_t<tcnn::network_precision_t, 4>*)dloss_doutput = local_dL_doutput;

		dloss_doutput += padded_output_width;
		network_output += padded_output_width;
	}

	if (exposure_gradient) {
		// Assume symmetric loss
		Array3f dloss_by_dgt = -lg.gradient / xy_pdf;

		if (!train_in_linear_colors) {
			dloss_by_dgt /= srgb_to_linear_derivative(rgbtarget);
		}

		// 2^exposure * log(2)
		Array3f dloss_by_dexposure = loss_scale * dloss_by_dgt * exposure_scale * 0.6931471805599453f;
		atomicAdd(&exposure_gradient[img].x(), dloss_by_dexposure.x());
		atomicAdd(&exposure_gradient[img].y(), dloss_by_dexposure.y());
		atomicAdd(&exposure_gradient[img].z(), dloss_by_dexposure.z());
	}

	if (compacted_numsteps == numsteps && envmap_gradient) {
		Array3f loss_gradient = lg.gradient;
		if (envmap_loss_type != loss_type) {
			loss_gradient = loss_and_gradient(rgbtarget, rgb_ray, envmap_loss_type).gradient;
		}

		Array3f dloss_by_dbackground = T * loss_gradient;
		if (!train_in_linear_colors) {
			dloss_by_dbackground /= srgb_to_linear_derivative(background_color);
		}

		tcnn::vector_t<tcnn::network_precision_t, 4> dL_denvmap;
		dL_denvmap[0] = loss_scale * dloss_by_dbackground.x();
		dL_denvmap[1] = loss_scale * dloss_by_dbackground.y();
		dL_denvmap[2] = loss_scale * dloss_by_dbackground.z();


		float dloss_by_denvmap_alpha = dloss_by_dbackground.matrix().dot(-pre_envmap_background_color.matrix());

		// dL_denvmap[3] = loss_scale * dloss_by_denvmap_alpha;
		dL_denvmap[3] = (tcnn::network_precision_t)0;

		deposit_envmap_gradient(dL_denvmap, envmap_gradient, envmap_resolution, dir);
	}
}

__global__ void compute_loss_kernel_train_nerf_distill(
	const uint32_t n_repidi,
	const uint32_t n_rays,
	BoundingBox aabb,
	const uint32_t n_rays_total,
	default_rng_t rng,
	const uint32_t max_samples_compacted,
	const uint32_t* __restrict__ rays_counter,
	float loss_scale,
	int padded_output_width,
	const float* __restrict__ envmap_data,
	float* __restrict__ envmap_gradient,
	const Vector2i envmap_resolution,
	ELossType envmap_loss_type,
	Array3f background_color,
	EColorSpace color_space,
	bool train_with_random_bg_color,
	bool train_in_linear_colors,
	const __half* __restrict__ training_images,
	const uint32_t n_training_images,
	Vector2i resolution,
	const tcnn::network_precision_t* network_output,
	uint32_t* __restrict__ numsteps_counter,
	const uint32_t* __restrict__ ray_indices_in,
	const Ray* __restrict__ rays_in,
	uint32_t* __restrict__ numsteps_in,
	PitchedPtr<const NerfCoordinate> coords_in,
	PitchedPtr<NerfCoordinate> coords_out,
	tcnn::network_precision_t* dloss_doutput,
	ELossType loss_type,
	float* __restrict__ loss_output,
	bool max_level_rand_training,
	float* __restrict__ max_level_compacted_ptr,
	ENerfActivation rgb_activation,
	ENerfActivation density_activation,
	bool snap_to_pixel_centers,
	float* __restrict__ error_map,
	const float* __restrict__ cdf_x_cond_y,
	const float* __restrict__ cdf_y,
	const float* __restrict__ cdf_img,
	const Vector2i error_map_res,
	const Vector2i error_map_cdf_res,
	const float* __restrict__ sharpness_data,
	Eigen::Vector2i sharpness_resolution,
	float* __restrict__ sharpness_grid,
	float* __restrict__ density_grid,
	const float* __restrict__ mean_density_ptr,
	const Eigen::Array3f* __restrict__ exposure,
	Eigen::Array3f* __restrict__ exposure_gradient,
	const float near_distance,
	Distiller** cage_infos,
	AuxSample* auxsamples
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= *rays_counter) { return; }

	// grab the number of samples for this ray, and the first sample
	uint32_t numsteps = numsteps_in[i * 2 + 0];
	uint32_t base = numsteps_in[i * 2 + 1];

	coords_in += base;
	network_output += base * padded_output_width;
	auxsamples += base;

	float T = 1.f;

	float EPSILON = 1e-4f;

	Array3f rgb_ray = Array3f::Zero();
	Vector3f hitpoint = Vector3f::Zero();

	bool emptying = auxsamples->emptying;

	uint32_t compacted_numsteps = 0;
	for (; compacted_numsteps < numsteps; ++compacted_numsteps) {
		if (T < EPSILON) {
			break;
		}

		const tcnn::vector_t<tcnn::network_precision_t, 4> local_network_output = *(tcnn::vector_t<tcnn::network_precision_t, 4>*)network_output;
		const Array3f rgb = network_to_rgb(local_network_output, rgb_activation);
		const Vector3f pos = unwarp_position(coords_in.ptr->pos.p, aabb);
		const float dt = unwarp_dt(coords_in.ptr->dt);

		float density = network_to_density(float(local_network_output[3]), density_activation);
		float alpha = 1.f - __expf(-density * dt);

		if (emptying) // we are emptying, only consider relevants
		{
			if (auxsamples->overwritten || auxsamples->outside)
				alpha = 0.0f;
		}
		else if (auxsamples->overwritten) // TODO: do it properly, from fallback network!
		{
			alpha = 0.0f;
		}

		const float weight = alpha * T;

		rgb_ray += weight * rgb;
		hitpoint += weight * pos;
		T *= (1.f - alpha);

		network_output += padded_output_width;
		coords_in += 1;
		auxsamples += 1;
	}
	hitpoint /= (1.0f - T);

	// Must be same seed as above to obtain the same
	// background color.
	uint32_t ray_idx = ray_indices_in[i];
	rng.advance(ray_idx * N_MAX_RANDOM_SAMPLES_PER_RAY());

	float img_pdf = 1.0f;
	uint32_t img = image_idx(ray_idx, n_rays, n_rays_total, n_training_images, cdf_img, &img_pdf);

	float xy_pdf = 1.0f;
	Vector2f xy = nerf_random_image_pos_training(rng, resolution, snap_to_pixel_centers, cdf_x_cond_y, cdf_y, error_map_cdf_res, img, &xy_pdf);
	float max_level = max_level_rand_training ? (random_val(rng) * 2.0f) : 1.0f; // Multiply by 2 to ensure 50% of training is at max level

	if (train_with_random_bg_color) {
		background_color = random_val_3d(rng);
	}
	Array3f pre_envmap_background_color = background_color = srgb_to_linear(background_color);

	// Composit background behind envmap
	Array4f envmap_value;
	Vector3f dir;
	if (envmap_data) {
		dir = rays_in[i].d;
		envmap_value = read_envmap(envmap_data, envmap_resolution, dir);
		background_color = envmap_value.head<3>() + background_color * (1.0f - envmap_value.w());
	}

	Array3f exposure_scale = (0.6931471805599453f * exposure[img]).exp();
	// Array3f rgbtarget = composit_and_lerp(xy, resolution, img, training_images, background_color, exposure_scale);
	// Array3f rgbtarget = composit(xy, resolution, img, training_images, background_color, exposure_scale);
	Array4f texsamp = read_rgba(xy, resolution, img, training_images);

	auto bg_old = background_color;
	Array3f rgbtarget;
	if (train_in_linear_colors || color_space == EColorSpace::Linear) {
		rgbtarget = exposure_scale * texsamp.head<3>() + (1.0f - texsamp.w()) * background_color;

		if (!train_in_linear_colors) {
			rgbtarget = linear_to_srgb(rgbtarget);
			background_color = linear_to_srgb(background_color);
		}
	}
	else if (color_space == EColorSpace::SRGB) {
		background_color = linear_to_srgb(background_color);
		if (texsamp.w() > 0) {
			rgbtarget = linear_to_srgb(exposure_scale * texsamp.head<3>() / texsamp.w()) * texsamp.w() + (1.0f - texsamp.w()) * background_color;
		}
		else {
			rgbtarget = background_color;
		}
	}

	if (emptying) // Our goal is to become see-through!
	{
		if (train_in_linear_colors || color_space == EColorSpace::Linear) {
			rgbtarget = bg_old;
			if (!train_in_linear_colors) {
				rgbtarget = linear_to_srgb(rgbtarget);
			}
		}
		else if (color_space == EColorSpace::SRGB) {
			rgbtarget = background_color;
		}
	}

	if (compacted_numsteps == numsteps) {
		// support arbitrary background colors
		rgb_ray += T * background_color;
	}

	// Step again, this time computing loss
	network_output -= padded_output_width * compacted_numsteps; // rewind the pointer
	coords_in -= compacted_numsteps;
	auxsamples -= compacted_numsteps;

	uint32_t compacted_base = atomicAdd(numsteps_counter, compacted_numsteps); // first entry in the array is a counter
	compacted_numsteps = min(max_samples_compacted - min(max_samples_compacted, compacted_base), compacted_numsteps);
	numsteps_in[i * 2 + 0] = compacted_numsteps;
	numsteps_in[i * 2 + 1] = compacted_base;
	if (compacted_numsteps == 0) {
		return;
	}

	max_level_compacted_ptr += compacted_base;
	coords_out += compacted_base;

	dloss_doutput += compacted_base * padded_output_width;

	LossAndGradient lg = loss_and_gradient(rgbtarget, rgb_ray, loss_type);
	lg.loss /= img_pdf * xy_pdf;

	// Note: dividing the gradient by the PDF would cause unbiased loss estimates.
	// Essentially: variance reduction, but otherwise the same optimization.
	// We _dont_ want that. If importance sampling is enabled, we _do_ actually want
	// to change the weighting of the loss function. So don't divide.
	// lg.gradient /= img_pdf * xy_pdf;

	float mean_loss = lg.loss.mean();
	if (loss_output) {
		loss_output[i] = mean_loss / (float)n_rays;
	}

	if (error_map) {
		const Vector2f pos = (xy.cwiseProduct(error_map_res.cast<float>()) - Vector2f::Constant(0.5f)).cwiseMax(0.0f).cwiseMin(error_map_res.cast<float>() - Vector2f::Constant(1.0f + 1e-4f));
		const Vector2i pos_int = pos.cast<int>();
		const Vector2f weight = pos - pos_int.cast<float>();

		Vector2i idx = pos_int.cwiseMin(resolution - Vector2i::Constant(2)).cwiseMax(0);

		auto deposit_val = [&](int x, int y, float val) {
			atomicAdd(&error_map[img * error_map_res.prod() + y * error_map_res.x() + x], val);
		};

		if (sharpness_data && aabb.contains(hitpoint)) {
			Vector2i sharpness_pos = xy.cwiseProduct(sharpness_resolution.cast<float>()).cast<int>().cwiseMax(0).cwiseMin(sharpness_resolution - Vector2i::Constant(1));
			float sharp = sharpness_data[img * sharpness_resolution.prod() + sharpness_pos.y() * sharpness_resolution.x() + sharpness_pos.x()] + 1e-6f;

			// The maximum value of positive floats interpreted in uint format is the same as the maximum value of the floats.
			float grid_sharp = __uint_as_float(atomicMax((uint32_t*)&cascaded_grid_at(hitpoint, sharpness_grid, mip_from_pos(hitpoint)), __float_as_uint(sharp)));
			grid_sharp = fmaxf(sharp, grid_sharp); // atomicMax returns the old value, so compute the new one locally.

			mean_loss *= fmaxf(sharp / grid_sharp, 0.01f);
		}

		deposit_val(idx.x(), idx.y(), (1 - weight.x()) * (1 - weight.y()) * mean_loss);
		deposit_val(idx.x() + 1, idx.y(), weight.x() * (1 - weight.y()) * mean_loss);
		deposit_val(idx.x(), idx.y() + 1, (1 - weight.x()) * weight.y() * mean_loss);
		deposit_val(idx.x() + 1, idx.y() + 1, weight.x() * weight.y() * mean_loss);
	}

	loss_scale /= n_rays;

	const float output_l2_reg = rgb_activation == ENerfActivation::Exponential ? 1e-4f : 0.0f;
	const float output_l1_reg_density = *mean_density_ptr < NERF_MIN_OPTICAL_THICKNESS() ? 1e-4f : 0.0f;

	// now do it again computing gradients
	Array3f rgb_ray2 = { 0.f,0.f,0.f };
	T = 1.f;
	Vector3f origin = rays_in[i].o;
	for (uint32_t j = 0; j < compacted_numsteps; ++j) {
		if (max_level_rand_training) {
			max_level_compacted_ptr[j] = max_level;
		}
		// Compact network inputs
		NerfCoordinate* coord_out = coords_out(j);
		const NerfCoordinate* coord_in = coords_in(j);
		coord_out->copy_with_optional_light_dir(*coord_in, coords_out.stride_in_bytes);
		float dt = unwarp_dt(coord_in->dt);
		const tcnn::vector_t<tcnn::network_precision_t, 4> local_network_output = *(tcnn::vector_t<tcnn::network_precision_t, 4>*)network_output;
		const Array3f rgb = network_to_rgb(local_network_output, rgb_activation);
		const float density = network_to_density(float(local_network_output[3]), density_activation);
		float alpha = 1.f - __expf(-density * dt);

		if (auxsamples[j].emptying) // we are emptying, only consider relevants
		{
			if (auxsamples[j].overwritten || auxsamples[j].outside)
				alpha = 0.0f;
		}
		else if (auxsamples[j].overwritten) // TODO: do it properly, from fallback network!
		{
			alpha = 0.0f;
		}

		const float weight = alpha * T;
		rgb_ray2 += weight * rgb;
		T *= (1.f - alpha);

		// we know the suffix of this ray compared to where we are up to. note the suffix depends on this step's alpha as suffix = (1-alpha)*(somecolor), so dsuffix/dalpha = -somecolor = -suffix/(1-alpha)
		const Array3f suffix = rgb_ray - rgb_ray2;
		const Array3f dloss_by_drgb = weight * lg.gradient;

		tcnn::vector_t<tcnn::network_precision_t, 4> local_dL_doutput;

		// chain rule to go from dloss/drgb to dloss/dmlp_output
		local_dL_doutput[0] = loss_scale * (dloss_by_drgb.x() * network_to_rgb_derivative(local_network_output[0], rgb_activation) + fmaxf(0.0f, output_l2_reg * (float)local_network_output[0])); // Penalize way too large color values
		local_dL_doutput[1] = loss_scale * (dloss_by_drgb.y() * network_to_rgb_derivative(local_network_output[1], rgb_activation) + fmaxf(0.0f, output_l2_reg * (float)local_network_output[1]));
		local_dL_doutput[2] = loss_scale * (dloss_by_drgb.z() * network_to_rgb_derivative(local_network_output[2], rgb_activation) + fmaxf(0.0f, output_l2_reg * (float)local_network_output[2]));

		float density_derivative = network_to_density_derivative(float(local_network_output[3]), density_activation);
		float dloss_by_dmlp = density_derivative * (
			dt * lg.gradient.matrix().dot((T * rgb - suffix).matrix())
			);

		//static constexpr float mask_supervision_strength = 1.f; // we are already 'leaking' mask information into the nerf via the random bg colors; setting this to eg between 1 and  100 encourages density towards 0 in such regions.
		//dloss_by_dmlp += (texsamp.w()<0.001f) ? mask_supervision_strength * weight : 0.f ;

		local_dL_doutput[3] =
			loss_scale * dloss_by_dmlp +
			(float(local_network_output[3]) < 0.0f ? -output_l1_reg_density : 0.0f) +
			(float(local_network_output[3]) > -10.0f && (unwarp_position(coord_in->pos.p, aabb) - origin).norm() < near_distance ? 1e-4f : 0.0f);

		if (auxsamples[j].overwritten || (auxsamples[j].emptying && auxsamples[j].outside))
		{
			local_dL_doutput[0] = 0.0f;
			local_dL_doutput[1] = 0.0f;
			local_dL_doutput[2] = 0.0f;
			local_dL_doutput[3] = 0.0f;
		}

		*(tcnn::vector_t<tcnn::network_precision_t, 4>*)dloss_doutput = local_dL_doutput;

		dloss_doutput += padded_output_width;
		network_output += padded_output_width;
	}

	if (exposure_gradient) {
		// Assume symmetric loss
		Array3f dloss_by_dgt = -lg.gradient / xy_pdf;

		if (!train_in_linear_colors) {
			dloss_by_dgt /= srgb_to_linear_derivative(rgbtarget);
		}

		// 2^exposure * log(2)
		Array3f dloss_by_dexposure = loss_scale * dloss_by_dgt * exposure_scale * 0.6931471805599453f;
		atomicAdd(&exposure_gradient[img].x(), dloss_by_dexposure.x());
		atomicAdd(&exposure_gradient[img].y(), dloss_by_dexposure.y());
		atomicAdd(&exposure_gradient[img].z(), dloss_by_dexposure.z());
	}

	if (compacted_numsteps == numsteps && envmap_gradient) {
		Array3f loss_gradient = lg.gradient;
		if (envmap_loss_type != loss_type) {
			loss_gradient = loss_and_gradient(rgbtarget, rgb_ray, envmap_loss_type).gradient;
		}

		Array3f dloss_by_dbackground = T * loss_gradient;
		if (!train_in_linear_colors) {
			dloss_by_dbackground /= srgb_to_linear_derivative(background_color);
		}

		tcnn::vector_t<tcnn::network_precision_t, 4> dL_denvmap;
		dL_denvmap[0] = loss_scale * dloss_by_dbackground.x();
		dL_denvmap[1] = loss_scale * dloss_by_dbackground.y();
		dL_denvmap[2] = loss_scale * dloss_by_dbackground.z();


		float dloss_by_denvmap_alpha = dloss_by_dbackground.matrix().dot(-pre_envmap_background_color.matrix());

		// dL_denvmap[3] = loss_scale * dloss_by_denvmap_alpha;
		dL_denvmap[3] = (tcnn::network_precision_t)0;

		deposit_envmap_gradient(dL_denvmap, envmap_gradient, envmap_resolution, dir);
	}
}

__global__ void compute_cam_gradient_train_nerf(
	const uint32_t n_rays,
	const uint32_t n_rays_total,
	default_rng_t rng,
	const BoundingBox aabb,
	const uint32_t* __restrict__ rays_counter,
	const TrainingXForm* training_xforms,
	Vector2i resolution,
	bool snap_to_pixel_centers,
	Vector3f* cam_pos_gradient,
	Vector3f* cam_rot_gradient,
	const uint32_t n_training_images,
	const uint32_t* __restrict__ ray_indices_in,
	const Ray* __restrict__ rays_in,
	uint32_t* __restrict__ numsteps_in,
	PitchedPtr<NerfCoordinate> coords,
	PitchedPtr<NerfCoordinate> coords_gradient,
	float* __restrict__ distortion_gradient,
	float* __restrict__ distortion_gradient_weight,
	const Vector2i distortion_resolution,
	Vector2f* cam_focal_length_gradient,
	const float* __restrict__ cdf_x_cond_y,
	const float* __restrict__ cdf_y,
	const float* __restrict__ cdf_img,
	const Vector2i error_map_res
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= *rays_counter) { return; }

	// grab the number of samples for this ray, and the first sample
	uint32_t numsteps = numsteps_in[i*2+0];
	if (numsteps == 0) {
		// The ray doesn't matter. So no gradient onto the camera
		return;
	}

	uint32_t base = numsteps_in[i*2+1];
	coords += base;
	coords_gradient += base;

	// Must be same seed as above to obtain the same
	// background color.
	uint32_t ray_idx = ray_indices_in[i];
	uint32_t img = image_idx(ray_idx, n_rays, n_rays_total, n_training_images, cdf_img);

	const Matrix<float, 3, 4>& xform = training_xforms[img].start;

	Ray ray = rays_in[i];
	Ray ray_gradient = { Vector3f::Zero(), Vector3f::Zero() };

	// Compute ray gradient
	for (uint32_t j = 0; j < numsteps; ++j) {
		// pos = ray.o + t * ray.d;

		const Vector3f warped_pos = coords(j)->pos.p;
		const Vector3f pos_gradient = coords_gradient(j)->pos.p.cwiseProduct(warp_position_derivative(warped_pos, aabb));
		ray_gradient.o += pos_gradient;
		const Vector3f pos = unwarp_position(warped_pos, aabb);

		// Scaled by t to account for the fact that further-away objects' position
		// changes more rapidly as the direction changes.
		float t = (pos - ray.o).norm();
		const Vector3f dir_gradient = coords_gradient(j)->dir.d.cwiseProduct(warp_direction_derivative(coords(j)->dir.d));
		ray_gradient.d += pos_gradient * t + dir_gradient;
	}

	// Projection of the raydir gradient onto the plane normal to raydir,
	// because that's the only degree of motion that the raydir has.
	ray_gradient.d -= ray.d * ray_gradient.d.dot(ray.d);

	rng.advance(ray_idx * N_MAX_RANDOM_SAMPLES_PER_RAY());
	float xy_pdf = 1.0f;
	Vector2f xy = nerf_random_image_pos_training(rng, resolution, snap_to_pixel_centers, cdf_x_cond_y, cdf_y, error_map_res, img, &xy_pdf);

	if (distortion_gradient) {
		// Rotate ray gradient to obtain image plane gradient.
		// This has the effect of projecting the (already projected) ray gradient from the
		// tangent plane of the sphere onto the image plane (which is correct!).
		Vector3f image_plane_gradient = xform.block<3,3>(0,0).inverse() * ray_gradient.d;

		// Splat the resulting 2D image plane gradient into the distortion params
		deposit_image_gradient<2>(image_plane_gradient.head<2>() / xy_pdf, distortion_gradient, distortion_gradient_weight, distortion_resolution, xy);
	}

	if (cam_pos_gradient) {
		// Atomically reduce the ray gradient into the xform gradient
		#pragma unroll
		for (uint32_t j = 0; j < 3; ++j) {
			atomicAdd(&cam_pos_gradient[img][j], ray_gradient.o[j] / xy_pdf);
		}
	}

	if (cam_rot_gradient) {
		// Rotation is averaged in log-space (i.e. by averaging angle-axes).
		// Due to our construction of ray_gradient.d, ray_gradient.d and ray.d are
		// orthogonal, leading to the angle_axis magnitude to equal the magnitude
		// of ray_gradient.d.
		Vector3f angle_axis = ray.d.cross(ray_gradient.d);

		// Atomically reduce the ray gradient into the xform gradient
		#pragma unroll
		for (uint32_t j = 0; j < 3; ++j) {
			atomicAdd(&cam_rot_gradient[img][j], angle_axis[j] / xy_pdf);
		}
	}
}

__global__ void shade_kernel_nerf(
	const uint32_t n_elements,
	Array4f* __restrict__ rgba,
	float* __restrict__ depth,
	Array3f* __restrict__ normals,
	NerfPayload* __restrict__ payloads,
	ERenderMode render_mode,
	// ERelightingDisplayMode relighting_display_mode,
	bool train_in_linear_colors,
	Array4f* __restrict__ frame_buffer,
	float* __restrict__ depth_buffer
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;
	NerfPayload& payload = payloads[i];

	Array4f tmp = rgba[i];

	if (render_mode == ERenderMode::Normals) {
		Array3f n = tmp.head<3>().matrix().normalized().array();
		tmp.head<3>() = (0.5f * n + Array3f::Constant(0.5f)) * tmp.w();
	} else if (render_mode == ERenderMode::Cost) {
		float col = (float)payload.n_steps / 128;
		tmp = {col, col, col, 1.0f};
	}

	if (!train_in_linear_colors && (render_mode == ERenderMode::Shade || render_mode == ERenderMode::Slice)) {
		// Accumulate in linear colors
		tmp.head<3>() = srgb_to_linear(tmp.head<3>());
	}

	frame_buffer[payload.idx] = tmp + frame_buffer[payload.idx] * (1.0f - tmp.w());
	if (render_mode != ERenderMode::Slice && tmp.w() > 0.2f) {
		depth_buffer[payload.idx] = depth[i];
	}
}

__global__ void compact_kernel_nerf(
	const uint32_t n_elements,
	Array4f* src_rgba, float* src_depth, Array3f* src_normal, NerfPayload* src_payloads,
	Array4f* dst_rgba, float* dst_depth, Array3f* dst_normal, NerfPayload* dst_payloads,
	Array4f* dst_final_rgba, float* dst_final_depth, Array3f* dst_final_normal, NerfPayload* dst_final_payloads,
	uint32_t* counter, uint32_t* finalCounter
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	NerfPayload& src_payload = src_payloads[i];

	if (src_payload.alive) {
		uint32_t idx = atomicAdd(counter, 1);
		dst_payloads[idx] = src_payload;
		dst_rgba[idx] = src_rgba[i];
		dst_depth[idx] = src_depth[i];
		dst_normal[idx] = src_normal[i];
	} else if (src_rgba[i].w() > 0.001f) {
		uint32_t idx = atomicAdd(finalCounter, 1);
		dst_final_payloads[idx] = src_payload;
		dst_final_rgba[idx] = src_rgba[i];
		dst_final_depth[idx] = src_depth[i];
		dst_final_normal[idx] = src_normal[i];
	}
}

__global__ void init_rays_with_payload_kernel_nerf(
	uint32_t sample_index,
	NerfPayload* __restrict__ payloads,
	Vector2i resolution,
	Vector2f focal_length,
	Matrix<float, 3, 4> camera_matrix0,
	Matrix<float, 3, 4> camera_matrix1,
	Vector4f rolling_shutter,
	Vector2f screen_center,
	bool snap_to_pixel_centers,
	BoundingBox aabb,
	float plane_z,
	float dof,
	CameraDistortion camera_distortion,
	const float* __restrict__ envmap_data,
	const Vector2i envmap_resolution,
	Array4f* __restrict__ framebuffer,
	float* __restrict__ depthbuffer,
	const float* __restrict__ distortion_data,
	const Vector2i distortion_resolution,
	ERenderMode render_mode
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	uint32_t idx = x + resolution.x() * y;

	if (plane_z < 0) {
		dof = 0.0;
	}

	// TODO: pixel_to_ray also immediately computes u,v for the pixel, so this is somewhat redundant
	float u = (x+0.5f) * (1.f / resolution.x());
	float v = (y+0.5f) * (1.f / resolution.y());
	float ray_time = rolling_shutter.x() + rolling_shutter.y() * u + rolling_shutter.z() * v + rolling_shutter.w() * ld_random_val(sample_index, idx * 72239731);
	Ray ray = pixel_to_ray(
		sample_index,
		{x, y},
		resolution,
		focal_length,
		camera_matrix0 * ray_time + camera_matrix1 * (1.f - ray_time),
		screen_center,
		snap_to_pixel_centers,
		plane_z,
		dof,
		camera_distortion,
		distortion_data,
		distortion_resolution
	);

	NerfPayload& payload = payloads[idx];
	payload.max_weight = 0.0f;

	if (plane_z < 0) {
		float n = ray.d.norm();
		payload.origin = ray.o;
		payload.dir = (1.0f/n) * ray.d;
		payload.t = -plane_z*n;
		payload.idx = idx;
		payload.n_steps = 0;
		payload.alive = false;
		depthbuffer[idx] = -plane_z;
		return;
	}

	depthbuffer[idx] = 1e10f;

	ray.d = ray.d.normalized();

	if (envmap_data) {
		framebuffer[idx] = read_envmap(envmap_data, envmap_resolution, ray.d);
	}

	float t = fmaxf(aabb.ray_intersect(ray.o, ray.d).x(), NERF_RENDERING_NEAR_DISTANCE()) + 1e-6f;

	if (!aabb.contains(ray.o + ray.d * t)) {
		payload.origin = ray.o;
		payload.alive = false;
		return;
	}

	if (render_mode == ERenderMode::Distortion) {
		if (distortion_data) {
			framebuffer[idx].head<2>() = read_image<2>(distortion_data, distortion_resolution, Vector2f((float)x + 0.5f, (float)y + 0.5f).cwiseQuotient(resolution.cast<float>())) * 50.0f + Vector2f::Constant(0.5f);
		} else {
			framebuffer[idx].head<2>() = Vector2f{0.5f, 0.5f};
		}
		framebuffer[idx].tail<2>() = Vector2f{0.5f, 1.0f};
		depthbuffer[idx] = 1.0f;
		payload.origin = ray.o + ray.d * 10000.0f;
		payload.alive = false;
		return;
	}

	payload.origin = ray.o;
	payload.dir = ray.d;
	payload.t = t;
	payload.idx = idx;
	payload.n_steps = 0;
	payload.alive = true;
}

static constexpr float MIN_PDF = 0.01f;

__global__ void construct_cdf_2d(
	uint32_t n_images,
	uint32_t height,
	uint32_t width,
	const float* __restrict__ data,
	float* __restrict__ cdf_x_cond_y,
	float* __restrict__ cdf_y
) {
	const uint32_t y = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t img = threadIdx.y + blockIdx.y * blockDim.y;
	if (y >= height || img >= n_images) return;

	const uint32_t offset_xy = img * height * width + y * width;
	data += offset_xy;
	cdf_x_cond_y += offset_xy;

	float cum = 0;
	for (uint32_t x = 0; x < width; ++x) {
		cum += data[x] + 1e-10f;
		cdf_x_cond_y[x] = cum;
	}

	cdf_y[img * height + y] = cum;
	float norm = __frcp_rn(cum);

	for (uint32_t x = 0; x < width; ++x) {
		cdf_x_cond_y[x] = (1.0f - MIN_PDF) * cdf_x_cond_y[x] * norm + MIN_PDF * (float)(x+1) / (float)width;
	}
}

__global__ void construct_cdf_1d(
	uint32_t n_images,
	uint32_t height,
	float* __restrict__ cdf_y,
	float* __restrict__ cdf_img
) {
	const uint32_t img = threadIdx.x + blockIdx.x * blockDim.x;
	if (img >= n_images) return;

	cdf_y += img * height;

	float cum = 0;
	for (uint32_t y = 0; y < height; ++y) {
		cum += cdf_y[y];
		cdf_y[y] = cum;
	}

	cdf_img[img] = cum;

	float norm = __frcp_rn(cum);
	for (uint32_t y = 0; y < height; ++y) {
		cdf_y[y] = (1.0f - MIN_PDF) * cdf_y[y] * norm + MIN_PDF * (float)(y+1) / (float)height;
	}
}

__global__ void safe_divide(const uint32_t num_elements, float* __restrict__ inout, const float* __restrict__ divisor) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= num_elements) return;

	float local_divisor = divisor[i];
	inout[i] = local_divisor > 0.0f ? (inout[i] / local_divisor) : 0.0f;
}

void Testbed::NerfTracer::init_rays_from_camera(
	uint32_t sample_index,
	uint32_t padded_output_width,
	uint32_t n_extra_dims,
	const Vector2i& resolution,
	const Vector2f& focal_length,
	const Matrix<float, 3, 4>& camera_matrix0,
	const Matrix<float, 3, 4>& camera_matrix1,
	const Vector4f& rolling_shutter,
	Vector2f screen_center,
	bool snap_to_pixel_centers,
	const BoundingBox& render_aabb,
	float plane_z,
	float dof,
	const CameraDistortion& camera_distortion,
	const float* envmap_data,
	const Vector2i& envmap_resolution,
	const float* distortion_data,
	const Vector2i& distortion_resolution,
	Eigen::Array4f* frame_buffer,
	float* depth_buffer,
	uint8_t *grid,
	int show_accel,
	float cone_angle_constant,
	ERenderMode render_mode,
	hipStream_t stream
) {
	// Make sure we have enough memory reserved to render at the requested resolution
	size_t n_pixels = (size_t)resolution.x() * resolution.y();
	enlarge(n_pixels, padded_output_width, n_extra_dims, stream);

	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)resolution.x(), threads.x), div_round_up((uint32_t)resolution.y(), threads.y), 1 };
	init_rays_with_payload_kernel_nerf<<<blocks, threads, 0, stream>>>(
		sample_index,
		m_rays[0].payload,
		resolution,
		focal_length,
		camera_matrix0,
		camera_matrix1,
		rolling_shutter,
		screen_center,
		snap_to_pixel_centers,
		render_aabb,
		plane_z,
		dof,
		camera_distortion,
		envmap_data,
		envmap_resolution,
		frame_buffer,
		depth_buffer,
		distortion_data,
		distortion_resolution,
		render_mode
	);

	m_n_rays_initialized = resolution.x() * resolution.y();

	CUDA_CHECK_THROW(hipMemsetAsync(m_rays[0].rgba, 0, m_n_rays_initialized * sizeof(Array4f), stream));
	CUDA_CHECK_THROW(hipMemsetAsync(m_rays[0].depth, 0, m_n_rays_initialized * sizeof(float), stream));
	CUDA_CHECK_THROW(hipMemsetAsync(m_rays[0].normal, 0, m_n_rays_initialized * sizeof(Array3f), stream));

	linear_kernel(advance_pos_nerf, 0, stream,
		m_n_rays_initialized,
		render_aabb,
		camera_matrix1.col(2),
		focal_length,
		sample_index,
		m_rays[0].payload,
		grid,
		(show_accel >= 0) ? show_accel : 0,
		cone_angle_constant
	);
}

template <typename T>
__global__ void clear_empty_space(
	const uint32_t n_elements,
	const bool* __restrict__ empty_mask,
    T* __restrict__ rgbsigma
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

    //if (empty_mask[i]) {
    //    rgbsigma[i] = -100.f;
    //}
}

uint32_t Testbed::NerfTracer::trace(
	NerfNetwork<network_precision_t>& network,
	const BoundingBox& render_aabb,
	const BoundingBox& train_aabb,
	const uint32_t n_training_images,
	const TrainingXForm* training_xforms,
	const Vector2f& focal_length,
	float cone_angle_constant,
	const uint8_t* grid,
	ERenderMode render_mode,
	const Eigen::Matrix<float, 3, 4> &camera_matrix,
	float depth_scale,
	int visualized_layer,
	int visualized_dim,
	ENerfActivation rgb_activation,
	ENerfActivation density_activation,
	int show_accel,
	float min_transmittance,
	float glow_y_cutoff,
	int glow_mode,
	const Eigen::Vector3f& light_dir,
	bool apply_operators,
	hipStream_t stream
) {
	if (m_n_rays_initialized == 0) {
		return 0;
	}

	CUDA_CHECK_THROW(hipMemsetAsync(m_hit_counter.data(), 0, sizeof(uint32_t), stream));

	uint32_t n_alive = m_n_rays_initialized;
	// m_n_rays_initialized = 0;

	uint32_t i = 1;
	uint32_t double_buffer_index = 0;

	std::chrono::duration<double, std::milli> ms_ray_prep;
	std::chrono::duration<double, std::milli> ms_ray_mapping;
	std::chrono::duration<double, std::milli> ms_inference;
	std::chrono::duration<double, std::milli> ms_compositing;
	while (i < MARCH_ITER) {
		RaysNerfSoa& rays_current = m_rays[(double_buffer_index + 1) % 2];
		RaysNerfSoa& rays_tmp = m_rays[double_buffer_index % 2];
		++double_buffer_index;

		// Compact rays that did not diverge yet
		{
			CUDA_CHECK_THROW(hipMemsetAsync(m_alive_counter.data(), 0, sizeof(uint32_t), stream));
			linear_kernel(compact_kernel_nerf, 0, stream,
				n_alive,
				rays_tmp.rgba, rays_tmp.depth, rays_tmp.normal, rays_tmp.payload,
				rays_current.rgba, rays_current.depth, rays_current.normal, rays_current.payload,
				m_rays_hit.rgba, m_rays_hit.depth, m_rays_hit.normal, m_rays_hit.payload,
				m_alive_counter.data(), m_hit_counter.data()
			);
			CUDA_CHECK_THROW(hipMemcpyAsync(&n_alive, m_alive_counter.data(), sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
			CUDA_CHECK_THROW(hipStreamSynchronize(stream));
		}

		if (n_alive == 0) {
			break;
		}

		uint32_t n_steps_between_compaction = tcnn::clamp(m_n_rays_initialized / n_alive, (uint32_t)MIN_STEPS_INBETWEEN_COMPACTION, (uint32_t)MAX_STEPS_INBETWEEN_COMPACTION);

		uint32_t extra_stride = network.n_extra_dims() * sizeof(float);
		PitchedPtr<NerfCoordinate> input_data((NerfCoordinate*)m_network_input, 1, 0, extra_stride);
		PitchedPtr<NerfCoordinate> gradient_data((NerfCoordinate*)m_network_gradient, 1, 0, extra_stride);
		auto t_ray_prep_start = std::chrono::high_resolution_clock::now();
		linear_kernel(generate_next_nerf_network_inputs, 0, stream,
			n_alive,
			render_aabb,
			train_aabb,
			focal_length,
			camera_matrix.col(2),
			rays_current.payload,
			input_data,
			n_steps_between_compaction,
			grid,
			(show_accel>=0) ? show_accel : 0,
			cone_angle_constant,
			light_dir
		);
		auto t_ray_prep_end = std::chrono::high_resolution_clock::now();
		ms_ray_prep += t_ray_prep_end - t_ray_prep_start;

		uint32_t n_elements = next_multiple(n_alive * n_steps_between_compaction, tcnn::batch_size_granularity);
		
		// Update Poisson residual
		// This needs to be done in "deformed" space otherwise artifacts due to the residual density with appear at initial position
		//CUDA_CHECK_THROW(hipMemsetAsync(m_sh_in_boundary, 0, n_elements * sizeof(SH9RGB), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_sh_boundary, 0, n_elements * sizeof(SH9RGB), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_density_out_boundary, 0, n_elements * sizeof(float), stream));
		//CUDA_CHECK_THROW(hipMemsetAsync(m_density_in_boundary, 0, n_elements * sizeof(float), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_density_residual_boundary, 0, n_elements * sizeof(float), stream));
		if (apply_operators && m_edit_operators.size() > 0) {
			for (int i = m_edit_operators.size() - 1; i >= 0; i--) {
				auto& edit_operator = m_edit_operators[i];
				edit_operator->compute_poisson_full_residuals(
					stream, 
					n_alive,
					rays_current.payload, 
					input_data,
					//m_sh_in_boundary,
					m_sh_boundary,
					//m_density_in_boundary,
					m_density_out_boundary,
					m_density_residual_boundary
				);
				CUDA_CHECK_THROW(hipStreamSynchronize(stream));
			}
		}		
		
		GPUMatrix<bool> empty_mask;

		bool* empty_mask_ptr = nullptr;


		GPUMatrix<float> positions_matrix_old((float*)m_network_input, (sizeof(NerfCoordinate) + extra_stride) / sizeof(float), n_elements);
		GPUMatrix<network_precision_t, RM> rgbsigma_matrix_old((network_precision_t*)m_network_output_old, network.padded_output_width(), n_elements);
		network.inference_mixed_precision(stream, positions_matrix_old, rgbsigma_matrix_old);

		
		auto t_ray_mapping_start = std::chrono::high_resolution_clock::now();
		if (apply_operators && m_edit_operators.size() > 0) {
			empty_mask = GPUMatrix<bool>(1, n_elements);
			CUDA_CHECK_THROW(hipMemsetAsync(empty_mask.data(), false, empty_mask.n_bytes(), stream));
			for (int i = m_edit_operators.size() - 1; i >= 0; i--) {
				auto& edit_operator = m_edit_operators[i];
				edit_operator->map_rays(stream, input_data, empty_mask, n_elements);
			}
			empty_mask_ptr = empty_mask.data();
		}
		auto t_ray_mapping_end = std::chrono::high_resolution_clock::now();
		ms_ray_mapping += t_ray_mapping_end - t_ray_mapping_start;

		GPUMatrix<float> positions_matrix((float*)m_network_input, (sizeof(NerfCoordinate) + extra_stride) / sizeof(float), n_elements);
		GPUMatrix<float> gradients_matrix((float*)m_network_gradient, (sizeof(NerfCoordinate) + extra_stride) / sizeof(float), n_elements);
		GPUMatrix<network_precision_t, RM> rgbsigma_matrix((network_precision_t*)m_network_output, network.padded_output_width(), n_elements);
		
		auto t_inference_start = std::chrono::high_resolution_clock::now();
		network.inference_mixed_precision(stream, positions_matrix, rgbsigma_matrix);
		auto t_inference_end = std::chrono::high_resolution_clock::now();
		ms_inference += t_inference_end - t_inference_start;

		// Clear the empty space (when performing displacement)
		if (apply_operators && m_edit_operators.size() > 0) {
			linear_kernel(clear_empty_space<network_precision_t>, 0, stream, n_elements, empty_mask.data(), rgbsigma_matrix.data() + 3 * n_elements);
		}

		// Only compute gradients if requested for relighting or to render normals
		if (render_mode == ERenderMode::Normals) {
			network.input_gradient(stream, 3, positions_matrix, gradients_matrix);
		} else if (render_mode == ERenderMode::EncodingVis) {
			network.visualize_activation(stream, visualized_layer, visualized_dim, positions_matrix, positions_matrix);
		}

		// Update the residuals when performing color MVC correction
		//CUDA_CHECK_THROW(hipMemsetAsync(m_sh_initial, 0, n_elements * sizeof(SH9RGB), stream));
		//CUDA_CHECK_THROW(hipMemsetAsync(m_sh_new, 0, n_elements * sizeof(SH9RGB), stream));
		CUDA_CHECK_THROW(hipStreamSynchronize(stream));
		/*if (apply_operators && m_edit_operators.size() > 0) {
			for (int i = m_edit_operators.size() - 1; i >= 0; i--) {
				auto& edit_operator = m_edit_operators[i];
				edit_operator->compute_interpolated_radiance(
					stream, 
					n_alive,
					rays_current.payload, 
					input_data,
					m_sh_initial,
					m_sh_new
				);
				CUDA_CHECK_THROW(hipStreamSynchronize(stream));
			}
		}*/

		

		auto t_compositing_start = std::chrono::high_resolution_clock::now();
		linear_kernel(composite_kernel_nerf, 0, stream,
			n_alive,
			n_elements,
			i,
			train_aabb,
			glow_y_cutoff,
			glow_mode,
			n_training_images,
			training_xforms,
			camera_matrix,
			focal_length,
			depth_scale,
			rays_current.rgba,
			rays_current.depth,
			rays_current.normal,
			rays_current.payload,
			input_data,
			gradient_data,
			m_network_output_old,
			m_network_output,
			m_sh_boundary,
			m_density_out_boundary,
			m_density_residual_boundary,
			network.padded_output_width(),
			n_steps_between_compaction,
			render_mode,
			grid,
			rgb_activation,
			density_activation,
			show_accel,
			min_transmittance,
			empty_mask_ptr,
			m_poisson_target
		);
		auto t_compositing_end = std::chrono::high_resolution_clock::now();
		ms_compositing += t_compositing_end - t_compositing_start;


		i += n_steps_between_compaction;
	}

	// DEBUG ONLY
	// std::cout << "Ray prep time: " << ms_ray_prep.count() << std::endl;
	// std::cout << "Ray mapping time: " << ms_ray_mapping.count() << std::endl;
	// std::cout << "Inference time: " << ms_inference.count() << std::endl;
	// std::cout << "Compositing time: " << ms_compositing.count() << std::endl;

	uint32_t n_hit;
	CUDA_CHECK_THROW(hipMemcpyAsync(&n_hit, m_hit_counter.data(), sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
	CUDA_CHECK_THROW(hipStreamSynchronize(stream));
	return n_hit;
}

void Testbed::NerfTracer::enlarge(size_t n_elements, uint32_t padded_output_width, uint32_t n_extra_dims, hipStream_t stream) {
	n_elements = next_multiple(n_elements, size_t(tcnn::batch_size_granularity));
	size_t num_floats = sizeof(NerfCoordinate) / 4 + n_extra_dims;
	auto scratch = allocate_workspace_and_distribute<
		Array4f, float, Array3f, NerfPayload, // m_rays[0]
		Array4f, float, Array3f, NerfPayload, // m_rays[1]
		Array4f, float, Array3f, NerfPayload, // m_rays_hit

		network_precision_t,
		float,
		float,
		SH9RGB,
		//SH9RGB,
		//SH9RGB,
		float,
		float,
		//float,
		//SH9RGB,
		network_precision_t
	>(
		stream, &m_scratch_alloc,
		n_elements, n_elements, n_elements, n_elements,
		n_elements, n_elements, n_elements, n_elements,
		n_elements, n_elements, n_elements, n_elements,
		n_elements * MAX_STEPS_INBETWEEN_COMPACTION * padded_output_width,
		n_elements * MAX_STEPS_INBETWEEN_COMPACTION * num_floats,
		n_elements * MAX_STEPS_INBETWEEN_COMPACTION * num_floats,
		n_elements * MAX_STEPS_INBETWEEN_COMPACTION,
		//n_elements * MAX_STEPS_INBETWEEN_COMPACTION,
		//n_elements * MAX_STEPS_INBETWEEN_COMPACTION,
		n_elements * MAX_STEPS_INBETWEEN_COMPACTION,
		n_elements * MAX_STEPS_INBETWEEN_COMPACTION,
		//n_elements * MAX_STEPS_INBETWEEN_COMPACTION,
		//n_elements * MAX_STEPS_INBETWEEN_COMPACTION,
		n_elements * MAX_STEPS_INBETWEEN_COMPACTION * padded_output_width
	);

	m_rays[0].set(std::get<0>(scratch), std::get<1>(scratch), std::get<2>(scratch), std::get<3>(scratch), n_elements);
	m_rays[1].set(std::get<4>(scratch), std::get<5>(scratch), std::get<6>(scratch), std::get<7>(scratch), n_elements);
	m_rays_hit.set(std::get<8>(scratch), std::get<9>(scratch), std::get<10>(scratch), std::get<11>(scratch), n_elements);

	//m_network_output = std::get<12>(scratch);
	//m_network_input = std::get<13>(scratch);
	//m_network_gradient = std::get<14>(scratch);
	//m_sh_initial = std::get<15>(scratch);
	//m_sh_new = std::get<16>(scratch);
	//m_sh_boundary = std::get<17>(scratch);
	//m_density_out_boundary = std::get<18>(scratch);
	//m_density_residual_boundary = std::get<19>(scratch);
	//m_density_in_boundary = std::get<20>(scratch);
	//m_sh_in_boundary = std::get<21>(scratch);
	//m_network_output_old = std::get<22>(scratch);

	m_network_output = std::get<12>(scratch);
	m_network_input = std::get<13>(scratch);
	m_network_gradient = std::get<14>(scratch);
	m_sh_boundary = std::get<15>(scratch);
	m_density_out_boundary = std::get<16>(scratch);
	m_density_residual_boundary = std::get<17>(scratch);
	m_network_output_old = std::get<18>(scratch);
}

void Testbed::render_nerf(NerfNetwork<network_precision_t>& network, CudaRenderBuffer& render_buffer, const Vector2i& max_res, const Vector2f& focal_length, const Matrix<float, 3, 4>& camera_matrix0, const Matrix<float, 3, 4>& camera_matrix1, const Vector4f& rolling_shutter, const Vector2f& screen_center, bool apply_operators, hipStream_t stream) {
	float plane_z = m_slice_plane_z + m_scale;
	if (m_render_mode == ERenderMode::Slice) {
		plane_z = -plane_z;
	}

	ERenderMode render_mode = m_visualized_dimension > -1 ? ERenderMode::EncodingVis : m_render_mode;

	ScopeGuard tmp_memory_guard{[&]() {
		m_nerf.tracer.clear();
	}};

	// Our motion vector code can't undo f-theta and grid distortions -- so don't render these if DLSS is enabled.
	bool render_opencv_camera_distortion = m_nerf.render_with_camera_distortion && (!render_buffer.dlss() || m_nerf.render_distortion.mode == ECameraDistortionMode::Iterative);
	bool render_grid_camera_distortion = m_nerf.render_with_camera_distortion && !render_buffer.dlss();

	m_nerf.tracer.init_rays_from_camera(
		render_buffer.spp(),
		network.padded_output_width(),
		network.n_extra_dims(),
		render_buffer.in_resolution(),
		focal_length,
		camera_matrix0,
		camera_matrix1,
		rolling_shutter,
		screen_center,
		m_snap_to_pixel_centers,
		m_render_aabb,
		plane_z,
		m_dof,
		render_opencv_camera_distortion ? m_nerf.render_distortion : CameraDistortion{},
		m_envmap.envmap->params_inference(),
		m_envmap.resolution,
		render_grid_camera_distortion ? m_distortion.map->params_inference() : nullptr,
		m_distortion.resolution,
		render_buffer.frame_buffer(),
		render_buffer.depth_buffer(),
		m_nerf.density_grid_bitfield.data(),
		m_nerf.show_accel,
		m_nerf.cone_angle_constant,
		render_mode,
		stream
	);

	uint32_t n_hit;
	if (m_render_mode == ERenderMode::Slice) {
		n_hit = m_nerf.tracer.n_rays_initialized();
	} else {
		float depth_scale = 1.0f / m_nerf.training.dataset.scale;
		n_hit = m_nerf.tracer.trace(
			network,
			m_render_aabb,
			m_aabb,
			m_nerf.training.n_images_for_training,
			m_nerf.training.transforms.data(),
			focal_length,
			m_nerf.cone_angle_constant,
			m_nerf.density_grid_bitfield.data(),
			render_mode,
			camera_matrix1,
			depth_scale,
			m_visualized_layer,
			m_visualized_dimension,
			m_nerf.rgb_activation,
			m_nerf.density_activation,
			m_nerf.show_accel,
			m_nerf.rendering_min_transmittance,
			m_nerf.m_glow_y_cutoff,
			m_nerf.m_glow_mode,
			m_nerf.light_dir.normalized(),
			apply_operators,
			stream
		);
	}
	RaysNerfSoa& rays_hit = m_render_mode == ERenderMode::Slice ? m_nerf.tracer.rays_init() : m_nerf.tracer.rays_hit();

	if (m_render_mode == ERenderMode::Slice) {
		// Store colors in the normal buffer
		uint32_t n_elements = next_multiple(n_hit, tcnn::batch_size_granularity);
		const uint32_t floats_per_coord = sizeof(NerfCoordinate) / sizeof(float) + network.n_extra_dims();
		const uint32_t extra_stride = network.n_extra_dims() * sizeof(float); // extra stride on top of base NerfCoordinate struct

		m_nerf.vis_input.enlarge(n_elements * floats_per_coord);
		m_nerf.vis_rgba.enlarge(n_elements);
		linear_kernel(generate_nerf_network_inputs_at_current_position, 0, stream, n_hit, m_aabb, rays_hit.payload, PitchedPtr<NerfCoordinate>((NerfCoordinate*)m_nerf.vis_input.data(), 1, 0, extra_stride), m_nerf.light_dir.normalized());

		GPUMatrix<float> positions_matrix((float*)m_nerf.vis_input.data(), floats_per_coord, n_elements);
		GPUMatrix<float> rgbsigma_matrix((float*)m_nerf.vis_rgba.data(), 4, n_elements);

		if (m_visualized_dimension == -1) {
			network.inference(stream, positions_matrix, rgbsigma_matrix);
			linear_kernel(compute_nerf_density, 0, stream, n_hit, m_nerf.vis_rgba.data(), m_nerf.rgb_activation, m_nerf.density_activation);
		} else {
			network.visualize_activation(stream, m_visualized_layer, m_visualized_dimension, positions_matrix, rgbsigma_matrix);
		}

		linear_kernel(shade_kernel_nerf, 0, stream,
			n_hit,
			m_nerf.vis_rgba.data(),
			nullptr,
			rays_hit.normal,
			rays_hit.payload,
			m_render_mode,
			// m_nerf.tracer.relighting_display_mode,
			m_nerf.training.linear_colors,
			render_buffer.frame_buffer(),
			render_buffer.depth_buffer()
		);
		return;
	}

	linear_kernel(shade_kernel_nerf, 0, stream,
		n_hit,
		rays_hit.rgba,
		rays_hit.depth,
		rays_hit.normal,
		rays_hit.payload,
		m_render_mode,
		// m_nerf.tracer.relighting_display_mode,
		m_nerf.training.linear_colors,
		render_buffer.frame_buffer(),
		render_buffer.depth_buffer()
	);

	if (render_mode == ERenderMode::Cost) {
		std::vector<NerfPayload> payloads_final_cpu(n_hit);
		CUDA_CHECK_THROW(hipMemcpyAsync(payloads_final_cpu.data(), rays_hit.payload, n_hit * sizeof(NerfPayload), hipMemcpyDeviceToHost, stream));
		CUDA_CHECK_THROW(hipStreamSynchronize(stream));

		size_t total_n_steps = 0;
		for (uint32_t i = 0; i < n_hit; ++i) {
			total_n_steps += payloads_final_cpu[i].n_steps;
		}
		tlog::info() << "Total steps per hit= " << total_n_steps << "/" << n_hit << " = " << ((float)total_n_steps/(float)n_hit);
	}
}

void Testbed::Nerf::Training::set_camera_intrinsics(int frame_idx, float fx, float fy, float cx, float cy, float k1, float k2, float p1, float p2) {
	if (frame_idx < 0 || frame_idx >= dataset.n_images) {
		return;
	}
	if (fx <= 0.f) fx = fy;
	if (fy <= 0.f) fy = fx;
	if (cx < 0.f) cx = -cx; else cx = cx / image_resolution.x();
	if (cy < 0.f) cy = -cy; else cy = cy / image_resolution.y();
	ECameraDistortionMode mode = (k1 || k2 || p1 || p2) ? ECameraDistortionMode::Iterative : ECameraDistortionMode::None;
	dataset.metadata[frame_idx] = { { mode, k1, k2, p1, p2 }, { cx, cy }, { fx, fy },  Eigen::Vector4f::Zero() };
	update_metadata(frame_idx, frame_idx + 1);
}

void Testbed::Nerf::Training::set_camera_extrinsics(int frame_idx, const Eigen::Matrix<float, 3, 4> &camera_to_world) {
	if (frame_idx < 0 || frame_idx >= dataset.n_images) {
		return;
	}

	dataset.xforms[frame_idx].start = dataset.xforms[frame_idx].end =dataset.nerf_matrix_to_ngp(camera_to_world);
	cam_rot_offset[frame_idx].reset_state();
	cam_pos_offset[frame_idx].reset_state();
	cam_exposure[frame_idx].reset_state();
	update_transforms(frame_idx, frame_idx + 1);
}

void Testbed::Nerf::Training::reset_camera_extrinsics() {
	for (auto&& opt : cam_rot_offset) {
		opt.reset_state();
	}

	for (auto&& opt : cam_pos_offset) {
		opt.reset_state();
	}

	for (auto&& opt : cam_exposure) {
		opt.reset_state();
	}
}

Eigen::Matrix<float, 3, 4> Testbed::Nerf::Training::get_camera_extrinsics(int frame_idx) {
	if (frame_idx < 0 || frame_idx >= dataset.n_images) {
		return Eigen::Matrix<float, 3, 4>::Identity();
	}
	return dataset.ngp_matrix_to_nerf(transforms[frame_idx].start);
}

void Testbed::Nerf::Training::update_metadata(int first, int last) {
	if (last < 0) {
		last = dataset.n_images;
	}

	if (last > dataset.n_images) {
		last = dataset.n_images;
	}

	int n = last - first;
	if (n <= 0) {
		return;
	}

	std::vector<TrainingImageMetadata> updated_metadata;
	updated_metadata.reserve(n);
	for (uint32_t i = 0; i < n; ++i) {
		updated_metadata.push_back(dataset.metadata[i + first]);
		updated_metadata.back().focal_length += cam_focal_length_offset.variable();
	}

	metadata_gpu.enlarge(last);
	CUDA_CHECK_THROW(hipMemcpy(metadata_gpu.data() + first, updated_metadata.data(), n * sizeof(TrainingImageMetadata), hipMemcpyHostToDevice));
}

void Testbed::Nerf::Training::update_transforms(int first, int last) {
	if (last < 0) {
		last=dataset.n_images;
	}

	if (last > dataset.n_images) {
		last = dataset.n_images;
	}

	int n = last - first;
	if (n <= 0) {
		return;
	}

	if (transforms.size() < last) {
		transforms.resize(last);
	}

	for (uint32_t i = 0; i < n; ++i) {
		auto xform = dataset.xforms[i + first];
		Vector3f rot = cam_rot_offset[i + first].variable();
		float angle = rot.norm();
		rot /= angle;

		if (angle > 0) {
			xform.start.block<3, 3>(0, 0) = AngleAxisf(angle, rot) * xform.start.block<3, 3>(0, 0);
			xform.end.block<3, 3>(0, 0) = AngleAxisf(angle, rot) * xform.end.block<3, 3>(0, 0);
		}

		xform.start.col(3) += cam_pos_offset[i + first].variable();
		xform.end.col(3) += cam_pos_offset[i + first].variable();
		transforms[i + first] = xform;
	}

	transforms_gpu.enlarge(last);
	CUDA_CHECK_THROW(hipMemcpy(transforms_gpu.data() + first, transforms.data() + first, n * sizeof(TrainingXForm), hipMemcpyHostToDevice));
}

void Testbed::create_empty_nerf_dataset(size_t n_images, Eigen::Vector2i image_resolution, int aabb_scale, bool is_hdr) {
	m_nerf.training.dataset = ngp::create_empty_nerf_dataset(n_images, image_resolution, aabb_scale, is_hdr);
	load_nerf();
	m_nerf.training.n_images_for_training = 0;
	m_training_data_available = true;
}

void Testbed::load_nerf() {
	if (!m_data_path.empty()) {
		std::vector<fs::path> json_paths;
		if (m_data_path.is_directory()) {
			for (const auto& path : fs::directory{m_data_path}) {
				if (path.is_file() && equals_case_insensitive(path.extension(), "json")) {
					json_paths.emplace_back(path);
				}
			}
		} else if (equals_case_insensitive(m_data_path.extension(), "msgpack")) {
			load_snapshot(m_data_path.str());
			set_train(false);
			return;
		} else if (equals_case_insensitive(m_data_path.extension(), "json")) {
			json_paths.emplace_back(m_data_path);
		} else {
			throw std::runtime_error{"NeRF data path must either be a json file or a directory containing json files."};
		}

		m_nerf.training.dataset = ngp::load_nerf(json_paths, m_nerf.sharpen);
	}

	m_nerf.rgb_activation = m_nerf.training.dataset.is_hdr ? ENerfActivation::Exponential : ENerfActivation::Logistic;

	m_nerf.training.image_resolution = m_nerf.training.dataset.image_resolution;
	m_nerf.training.n_images_for_training = (int)m_nerf.training.dataset.n_images;

	m_nerf.training.update_metadata();

	m_nerf.training.cam_pos_gradient.resize(m_nerf.training.dataset.n_images, Vector3f::Zero());
	m_nerf.training.cam_pos_gradient_gpu.resize_and_copy_from_host(m_nerf.training.cam_pos_gradient);

	m_nerf.training.cam_exposure.resize(m_nerf.training.dataset.n_images, AdamOptimizer<Array3f>(1e-3f));
	m_nerf.training.cam_pos_offset.resize(m_nerf.training.dataset.n_images, AdamOptimizer<Vector3f>(1e-4f));
	m_nerf.training.cam_rot_offset.resize(m_nerf.training.dataset.n_images, RotationAdamOptimizer(1e-4f));
	m_nerf.training.cam_focal_length_offset = AdamOptimizer<Vector2f>(1e-5f);

	m_nerf.training.cam_rot_gradient.resize(m_nerf.training.dataset.n_images, Vector3f::Zero());
	m_nerf.training.cam_rot_gradient_gpu.resize_and_copy_from_host(m_nerf.training.cam_rot_gradient);

	m_nerf.training.cam_exposure_gradient.resize(m_nerf.training.dataset.n_images, Array3f::Zero());
	m_nerf.training.cam_exposure_gpu.resize_and_copy_from_host(m_nerf.training.cam_exposure_gradient);
	m_nerf.training.cam_exposure_gradient_gpu.resize_and_copy_from_host(m_nerf.training.cam_exposure_gradient);

	m_nerf.training.cam_focal_length_gradient = Vector2f::Zero();
	m_nerf.training.cam_focal_length_gradient_gpu.resize_and_copy_from_host(&m_nerf.training.cam_focal_length_gradient, 1);

	if (m_nerf.training.dataset.rays_data.data()) {
		m_nerf.training.near_distance = 0.0f;
		// m_nerf.training.optimize_exposure = true;
	}

	// Uncomment the following line to see how the network learns distortion from scratch rather than
	// starting from the distortion that's described by the training data.
	// m_nerf.training.dataset.camera_distortion = {};

	// Perturbation of the training cameras -- for debugging the online extrinsics learning code
	float perturb_amount = 0.0f;
	if (perturb_amount > 0.f) {
		for (uint32_t i = 0; i < m_nerf.training.dataset.n_images; ++i) {
			Vector3f rot = random_val_3d(m_rng) * perturb_amount;
			float angle = rot.norm();
			rot /= angle;
			auto trans = random_val_3d(m_rng);
			m_nerf.training.dataset.xforms[i].start.block<3,3>(0,0) = AngleAxisf(angle, rot).matrix() * m_nerf.training.dataset.xforms[i].start.block<3,3>(0,0);
			m_nerf.training.dataset.xforms[i].start.col(3) += trans * perturb_amount;
			m_nerf.training.dataset.xforms[i].end.block<3,3>(0,0) = AngleAxisf(angle, rot).matrix() * m_nerf.training.dataset.xforms[i].end.block<3,3>(0,0);
			m_nerf.training.dataset.xforms[i].end.col(3) += trans * perturb_amount;
		}
	}

	m_nerf.training.update_transforms();

	if (!m_nerf.training.dataset.metadata.empty()) {
		m_nerf.render_distortion = m_nerf.training.dataset.metadata[0].camera_distortion;
		m_screen_center = Eigen::Vector2f::Constant(1.f) - m_nerf.training.dataset.metadata[0].principal_point;
	}

	if (!is_pot(m_nerf.training.dataset.aabb_scale)) {
		throw std::runtime_error{std::string{"NeRF dataset's `aabb_scale` must be a power of two, but is "} + std::to_string(m_nerf.training.dataset.aabb_scale)};
	}

	int max_aabb_scale = 1 << (NERF_CASCADES()-1);
	if (m_nerf.training.dataset.aabb_scale > max_aabb_scale) {
		throw std::runtime_error{
			std::string{"NeRF dataset must have `aabb_scale <= "} + std::to_string(max_aabb_scale) +
			"`, but is " + std::to_string(m_nerf.training.dataset.aabb_scale) +
			". You can increase this limit by factors of 2 by incrementing `NERF_CASCADES()` and re-compiling."
		};
	}

	m_aabb = BoundingBox{Vector3f::Constant(0.5f), Vector3f::Constant(0.5f)};
	m_aabb.inflate(0.5f * std::min(1 << (NERF_CASCADES()-1), m_nerf.training.dataset.aabb_scale));
	m_raw_aabb = m_aabb;
	m_render_aabb = m_aabb;
	if (!m_nerf.training.dataset.render_aabb.is_empty()) {
		m_render_aabb = m_nerf.training.dataset.render_aabb.intersection(m_aabb);
	}

	m_nerf.max_cascade = 0;
	while ((1 << m_nerf.max_cascade) < m_nerf.training.dataset.aabb_scale) {
		++m_nerf.max_cascade;
	}

	// Perform fixed-size stepping in unit-cube scenes (like original NeRF) and exponential
	// stepping in larger scenes.
	m_nerf.cone_angle_constant = m_nerf.training.dataset.aabb_scale <= 1 ? 0.0f : (1.0f / 256.0f);

	m_up_dir = m_nerf.training.dataset.up;
}

void Testbed::update_density_grid_nerf(float decay, uint32_t n_uniform_density_grid_samples, uint32_t n_nonuniform_density_grid_samples, hipStream_t stream) {
	const uint32_t n_elements = NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_CASCADES();

	m_nerf.density_grid.enlarge(n_elements);

	const uint32_t n_density_grid_samples = n_uniform_density_grid_samples + n_nonuniform_density_grid_samples;

	const uint32_t padded_output_width = m_nerf_network->padded_density_output_width();

	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<
		NerfPosition,       // positions at which the NN will be queried for density evaluation
		uint32_t,           // indices of corresponding density grid cells
		float,              // the resulting densities `density_grid_tmp` to be merged with the running estimate of the grid
		network_precision_t // output of the MLP before being converted to densities.
	>(stream, &alloc, n_density_grid_samples, n_elements, n_elements, n_density_grid_samples * padded_output_width);

	NerfPosition* density_grid_positions = std::get<0>(scratch);
	uint32_t* density_grid_indices = std::get<1>(scratch);
	float* density_grid_tmp = std::get<2>(scratch);
	network_precision_t* mlp_out = std::get<3>(scratch);

	if (m_training_step == 0 || m_nerf.training.n_images_for_training != m_nerf.training.n_images_for_training_prev) {
		m_nerf.training.n_images_for_training_prev = m_nerf.training.n_images_for_training;
		if (m_training_step == 0) {
			m_nerf.density_grid_ema_step = 0;
		}
		// Only cull away empty regions where no camera is looking when the cameras are actually meaningful.
		if (!m_nerf.training.dataset.rays_data.data()) {
			linear_kernel(mark_untrained_density_grid, 0, stream, n_elements, m_nerf.density_grid.data(),
				m_nerf.training.n_images_for_training,
				m_nerf.training.metadata_gpu.data(),
				m_nerf.training.transforms_gpu.data(),
				m_nerf.training.image_resolution,
				m_training_step == 0
			);
		} else {
			CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.density_grid.data(), 0, sizeof(float)*n_elements, stream));
		}
	}

	uint32_t n_steps = 1;
	for (uint32_t i = 0; i < n_steps; ++i) {
		CUDA_CHECK_THROW(hipMemsetAsync(density_grid_tmp, 0, sizeof(float)*n_elements, stream));

		linear_kernel(generate_grid_samples_nerf_nonuniform, 0, stream,
			n_uniform_density_grid_samples,
			m_rng,
			m_nerf.density_grid_ema_step,
			m_aabb,
			m_nerf.density_grid.data(),
			density_grid_positions,
			density_grid_indices,
			m_nerf.max_cascade+1,
			-0.01f
		);
		m_rng.advance();

		linear_kernel(generate_grid_samples_nerf_nonuniform, 0, stream,
			n_nonuniform_density_grid_samples,
			m_rng,
			m_nerf.density_grid_ema_step,
			m_aabb,
			m_nerf.density_grid.data(),
			density_grid_positions+n_uniform_density_grid_samples,
			density_grid_indices+n_uniform_density_grid_samples,
			m_nerf.max_cascade+1,
			NERF_MIN_OPTICAL_THICKNESS()
		);
		m_rng.advance();

		GPUMatrix<network_precision_t, RM> density_matrix(mlp_out, padded_output_width, n_density_grid_samples);
		GPUMatrix<float> density_grid_position_matrix((float*)density_grid_positions, sizeof(NerfPosition)/sizeof(float), n_density_grid_samples);
		m_nerf_network->density(stream, density_grid_position_matrix, density_matrix, false);

		linear_kernel(splat_grid_samples_nerf_max_nearest_neighbor, 0, stream, n_density_grid_samples, density_grid_indices, mlp_out, density_grid_tmp, m_nerf.rgb_activation, m_nerf.density_activation);
		linear_kernel(ema_grid_samples_nerf, 0, stream, n_elements, decay, m_nerf.density_grid_ema_step, m_nerf.density_grid.data(), density_grid_tmp);

		++m_nerf.density_grid_ema_step;
	}

	update_density_grid_mean_and_bitfield(stream);
}

void Testbed::update_density_grid_nerf_render(uint32_t n_iterations, bool reset_grid, hipStream_t stream) {
		// Update the density grid with the new transformation
		for (int i = 0; i < n_iterations; i++) {
			update_density_grid_nerf_operator(NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE()*(m_nerf.max_cascade+1), 0, (reset_grid && i == 0), m_inference_stream);
		}
		CUDA_CHECK_THROW(hipStreamSynchronize(m_inference_stream));
}

__global__ void activate_network_density(
	const uint32_t n_elements,
	network_precision_t* network_output,
	ENerfActivation density_activation
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	network_output[i] = (network_precision_t)network_to_density(float(network_output[i]), density_activation);
}

void Testbed::update_density_grid_nerf_operator(uint32_t n_uniform_density_grid_samples, uint32_t n_nonuniform_density_grid_samples, bool reset_grid, hipStream_t stream) {
	float decay = m_nerf.training.density_grid_decay;

	const uint32_t n_elements = NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_CASCADES();

	m_nerf.density_grid.enlarge(n_elements);

	const uint32_t n_density_grid_samples = n_uniform_density_grid_samples + n_nonuniform_density_grid_samples;

	const uint32_t padded_output_width = m_nerf_network->padded_density_output_width();

	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<
		NerfPosition,       // positions at which the NN will be queried for density evaluation
		uint32_t,           // indices of corresponding density grid cells
		float,              // the resulting densities `density_grid_tmp` to be merged with the running estimate of the grid
		network_precision_t // output of the MLP before being converted to densities.
	>(stream, &alloc, n_density_grid_samples, n_elements, n_elements, n_density_grid_samples * padded_output_width);

	NerfPosition* density_grid_positions = std::get<0>(scratch);
	uint32_t* density_grid_indices = std::get<1>(scratch);
	float* density_grid_tmp = std::get<2>(scratch);
	network_precision_t* mlp_out = std::get<3>(scratch);

	if (reset_grid) {
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.density_grid.data(), 0, sizeof(float)*n_elements, stream));
	}

	uint32_t n_steps = 1;
	for (uint32_t i = 0; i < n_steps; ++i) {
		CUDA_CHECK_THROW(hipMemsetAsync(density_grid_tmp, 0, sizeof(float)*n_elements, stream));

		linear_kernel(generate_grid_samples_nerf_nonuniform, 0, stream,
			n_uniform_density_grid_samples,
			m_rng,
			m_nerf.density_grid_ema_step,
			m_aabb,
			m_nerf.density_grid.data(),
			density_grid_positions,
			density_grid_indices,
			m_nerf.max_cascade+1,
			-0.01f
		);
		m_rng.advance();

		linear_kernel(generate_grid_samples_nerf_nonuniform, 0, stream,
			n_nonuniform_density_grid_samples,
			m_rng,
			m_nerf.density_grid_ema_step,
			m_aabb,
			m_nerf.density_grid.data(),
			density_grid_positions+n_uniform_density_grid_samples,
			density_grid_indices+n_uniform_density_grid_samples,
			m_nerf.max_cascade+1,
			NERF_MIN_OPTICAL_THICKNESS()
		);
		m_rng.advance();

		GPUMatrix<bool> empty_mask;
		if (m_nerf.tracer.edit_operators().size() > 0) {
			empty_mask = GPUMatrix<bool>(1, n_elements);
			CUDA_CHECK_THROW(hipMemsetAsync(empty_mask.data(), false, empty_mask.n_bytes(), stream));
			for (int i = m_nerf.tracer.edit_operators().size() - 1; i >= 0; i--) {
				auto& edit_operator = m_nerf.tracer.edit_operators()[i];
				edit_operator->map_positions(stream, PitchedPtr<NerfPosition>((NerfPosition*)density_grid_positions, 1), empty_mask, n_density_grid_samples);
			}
		}

		GPUMatrix<network_precision_t, RM> density_matrix(mlp_out, padded_output_width, n_density_grid_samples);
		GPUMatrix<float> density_grid_position_matrix((float*)density_grid_positions, sizeof(NerfPosition)/sizeof(float), n_density_grid_samples);
		m_nerf_network->density(stream, density_grid_position_matrix, density_matrix, false);

		if (m_nerf.tracer.edit_operators().size() > 0) {
			linear_kernel(clear_empty_space<network_precision_t>, 0, stream, n_elements, empty_mask.data(), density_matrix.data());
		}

		// Pre-activate density in order to sum with the residual in density space!
		linear_kernel(activate_network_density, 0, stream, n_density_grid_samples, mlp_out, m_nerf.density_activation);

		if (m_nerf.tracer.edit_operators().size() > 0) {
			for (int i = m_nerf.tracer.edit_operators().size() - 1; i >= 0; i--) {
				auto& edit_operator = m_nerf.tracer.edit_operators()[i];
				edit_operator->compute_poisson_residual_density(stream,
					n_elements,
					PitchedPtr<NerfPosition>((NerfPosition*)density_grid_positions, 1),
					mlp_out);
			}
		}

		//if (m_nerf.tracer.edit_operators().size() > 0) {
		//	for (int i = m_nerf.tracer.edit_operators().size() - 1; i >= 0; i--) {
		//		auto& edit_operator = m_nerf.tracer.edit_operators()[i];
		//		edit_operator->kill_empty_density(stream,
		//			n_elements,
		//			PitchedPtr<NerfPosition>((NerfPosition*)density_grid_positions, 1),
		//			empty_mask,
		//			mlp_out);
		//	}
		//}

		linear_kernel(splat_grid_samples_nerf_max_nearest_neighbor_already_activated, 0, stream, n_density_grid_samples, density_grid_indices, mlp_out, density_grid_tmp);
		linear_kernel(ema_grid_samples_nerf, 0, stream, n_elements, decay, m_nerf.density_grid_ema_step, m_nerf.density_grid.data(), density_grid_tmp);

		++m_nerf.density_grid_ema_step;
	}

	update_density_grid_mean_and_bitfield(stream);
}

void Testbed::update_density_grid_mean_and_bitfield(hipStream_t stream) {
	const uint32_t n_elements = NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE();

	size_t size_including_mips = grid_mip_offset(NERF_CASCADES())/8;
	m_nerf.density_grid_bitfield.enlarge(size_including_mips);
	m_nerf.density_grid_mean.enlarge(reduce_sum_workspace_size(n_elements));

	CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.density_grid_mean.data(), 0, sizeof(float), stream));
	reduce_sum(m_nerf.density_grid.data(), [n_elements] __device__ (float val) { return fmaxf(val, 0.f) / (n_elements); }, m_nerf.density_grid_mean.data(), n_elements, stream);

	linear_kernel(grid_to_bitfield, 0, stream, n_elements/8 * NERF_CASCADES(), m_nerf.density_grid.data(), m_nerf.density_grid_bitfield.data(), m_nerf.density_grid_mean.data());

	for (uint32_t level = 1; level < NERF_CASCADES(); ++level) {
		linear_kernel(bitfield_max_pool, 0, stream, n_elements/64, m_nerf.get_density_grid_bitfield_mip(level-1), m_nerf.get_density_grid_bitfield_mip(level));
	}
}

void Testbed::Nerf::Training::Counters::prepare_for_training_steps(uint32_t n_training_steps, hipStream_t stream) {
	numsteps_counter.enlarge(n_training_steps);
	numsteps_counter_compacted.enlarge(n_training_steps);
	loss.enlarge(rays_per_batch * n_training_steps);
	CUDA_CHECK_THROW(hipMemsetAsync(numsteps_counter.data(), 0, sizeof(uint32_t)*n_training_steps, stream)); // clear the counter in the first slot
	CUDA_CHECK_THROW(hipMemsetAsync(numsteps_counter_compacted.data(), 0, sizeof(uint32_t)*n_training_steps, stream)); // clear the counter in the first slot
	CUDA_CHECK_THROW(hipMemsetAsync(loss.data(), 0, sizeof(float)*rays_per_batch*n_training_steps, stream));
}

float Testbed::Nerf::Training::Counters::update_after_training(uint32_t target_batch_size, uint32_t n_training_steps, hipStream_t stream) {
	std::vector<uint32_t> counter_cpu(n_training_steps);
	std::vector<uint32_t> compacted_counter_cpu(n_training_steps);
	numsteps_counter.copy_to_host(counter_cpu, n_training_steps);
	numsteps_counter_compacted.copy_to_host(compacted_counter_cpu, n_training_steps);
	measured_batch_size = 0;
	measured_batch_size_before_compaction = 0;

	for (uint32_t i = 0; i < n_training_steps; ++i) {
		if (counter_cpu[i] == 0 || compacted_counter_cpu[i] == 0) {
			return -1.f;
		}

		measured_batch_size_before_compaction += counter_cpu[i];
		measured_batch_size += compacted_counter_cpu[i];
	}

	measured_batch_size_before_compaction /= n_training_steps;
	measured_batch_size /= n_training_steps;

	float loss_scalar = reduce_sum(loss.data(), rays_per_batch * n_training_steps, stream) / (float)(n_training_steps);
	loss_scalar *= (float)measured_batch_size / (float)target_batch_size;

	rays_per_batch = (uint32_t)((float)rays_per_batch * (float)target_batch_size / (float)measured_batch_size);
	rays_per_batch = std::min(next_multiple(rays_per_batch, tcnn::batch_size_granularity), 1u << 18);

	return loss_scalar;
}

void Testbed::train_nerf(uint32_t target_batch_size, uint32_t n_training_steps, bool distill, hipStream_t stream) {
	if (m_nerf.training.include_sharpness_in_error) {
		size_t n_cells = NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_CASCADES();
		if (m_nerf.training.sharpness_grid.size() < n_cells) {
			m_nerf.training.sharpness_grid.enlarge(NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_CASCADES());
			CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.sharpness_grid.data(), 0, m_nerf.training.sharpness_grid.get_bytes(), stream));
		}

		if (m_training_step == 0) {
			CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.sharpness_grid.data(), 0, m_nerf.training.sharpness_grid.get_bytes(), stream));
		} else {
			linear_kernel(decay_sharpness_grid_nerf, 0, stream, m_nerf.training.sharpness_grid.size(), 0.95f, m_nerf.training.sharpness_grid.data());
		}
	}
	m_nerf.training.counters_rgb.prepare_for_training_steps(n_training_steps, stream);


	if (m_nerf.training.n_steps_since_cam_update == 0) {
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.cam_pos_gradient_gpu.data(), 0, m_nerf.training.cam_pos_gradient_gpu.get_bytes(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.cam_rot_gradient_gpu.data(), 0, m_nerf.training.cam_rot_gradient_gpu.get_bytes(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.cam_exposure_gradient_gpu.data(), 0, m_nerf.training.cam_exposure_gradient_gpu.get_bytes(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_distortion.map->gradients(), 0, sizeof(float)*m_distortion.map->n_params(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_distortion.map->gradient_weights(), 0, sizeof(float)*m_distortion.map->n_params(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.cam_focal_length_gradient_gpu.data(), 0, m_nerf.training.cam_focal_length_gradient_gpu.get_bytes(), stream));
	}


	if (m_nerf.training.n_steps_since_error_map_update == 0) {
		uint32_t n_samples_per_image = (m_nerf.training.n_steps_between_error_map_updates * m_nerf.training.counters_rgb.rays_per_batch) / m_nerf.training.dataset.n_images;
		m_nerf.training.error_map.resolution = Vector2i::Constant((int)(std::sqrt(std::sqrt((float)n_samples_per_image)) * 3.5f)).cwiseMin(m_nerf.training.image_resolution);
		m_nerf.training.error_map.data.resize(m_nerf.training.error_map.resolution.prod() * m_nerf.training.dataset.n_images);
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.error_map.data.data(), 0, m_nerf.training.error_map.data.get_bytes(), stream));
	}

	float* envmap_gradient = m_nerf.training.train_envmap ? m_envmap.envmap->gradients() : nullptr;
	if (envmap_gradient) {
		CUDA_CHECK_THROW(hipMemsetAsync(envmap_gradient, 0, sizeof(float)*m_envmap.envmap->n_params(), stream));
	}

	for (uint32_t i = 0; i < n_training_steps; ++i)
	{
		if(distill)
		{
			train_nerf_step_distill(
				target_batch_size,
				m_nerf.training.counters_rgb.rays_per_batch,
				m_nerf.training.counters_rgb.numsteps_counter.data() + i,
				m_nerf.training.counters_rgb.numsteps_counter_compacted.data() + i,
				m_nerf.training.counters_rgb.loss.data() + i * m_nerf.training.counters_rgb.rays_per_batch,
				m_training_stream
			);
		}
		else
		{
			train_nerf_step(
				target_batch_size,
				m_nerf.training.counters_rgb.rays_per_batch,
				m_nerf.training.counters_rgb.numsteps_counter.data() + i,
				m_nerf.training.counters_rgb.numsteps_counter_compacted.data() + i,
				m_nerf.training.counters_rgb.loss.data() + i * m_nerf.training.counters_rgb.rays_per_batch,
				m_training_stream
			);
		}

		m_trainer->optimizer_step(stream, LOSS_SCALE);

		++m_training_step;
	}

	if (envmap_gradient) {
		m_envmap.trainer->optimizer_step(stream, LOSS_SCALE*(float)n_training_steps);
	}
	m_loss_scalar = m_nerf.training.counters_rgb.update_after_training(target_batch_size, n_training_steps, stream);
	if (m_loss_scalar < 0.f) {
		m_loss_scalar = 0.f;
		tlog::warning() << "Nerf training generated 0 samples. Aborting training.";
		m_train = false;
	}

	update_loss_graph();

	// Compute CDFs from the error map
	m_nerf.training.n_steps_since_error_map_update += n_training_steps;
	// This is low-overhead enough to warrant always being on.
	// It makes for useful visualizations of the training error.
	bool accumulate_error = true;
	if (accumulate_error && m_nerf.training.n_steps_since_error_map_update >= m_nerf.training.n_steps_between_error_map_updates) {
		m_nerf.training.error_map.cdf_resolution = m_nerf.training.error_map.resolution;
		m_nerf.training.error_map.cdf_x_cond_y.resize(m_nerf.training.error_map.cdf_resolution.prod() * m_nerf.training.dataset.n_images);
		m_nerf.training.error_map.cdf_y.resize(m_nerf.training.error_map.cdf_resolution.y() * m_nerf.training.dataset.n_images);
		m_nerf.training.error_map.cdf_img.resize(m_nerf.training.dataset.n_images);

		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.error_map.cdf_x_cond_y.data(), 0, m_nerf.training.error_map.cdf_x_cond_y.get_bytes(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.error_map.cdf_y.data(), 0, m_nerf.training.error_map.cdf_y.get_bytes(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.error_map.cdf_img.data(), 0, m_nerf.training.error_map.cdf_img.get_bytes(), stream));

		const dim3 threads = { 16, 8, 1 };
		const dim3 blocks = { div_round_up((uint32_t)m_nerf.training.error_map.cdf_resolution.y(), threads.x), div_round_up((uint32_t)m_nerf.training.dataset.n_images, threads.y), 1 };
		construct_cdf_2d<<<blocks, threads, 0, stream>>>(
			m_nerf.training.dataset.n_images, m_nerf.training.error_map.cdf_resolution.y(), m_nerf.training.error_map.cdf_resolution.x(),
			m_nerf.training.error_map.data.data(),
			m_nerf.training.error_map.cdf_x_cond_y.data(),
			m_nerf.training.error_map.cdf_y.data()
		);
		linear_kernel(construct_cdf_1d, 0, stream,
			m_nerf.training.dataset.n_images,
			m_nerf.training.error_map.cdf_resolution.y(),
			m_nerf.training.error_map.cdf_y.data(),
			m_nerf.training.error_map.cdf_img.data()
		);

		// Compute image CDF on the CPU. It's single-threaded anyway. No use parallelizing.
		m_nerf.training.error_map.pmf_img_cpu.resize(m_nerf.training.error_map.cdf_img.size());
		m_nerf.training.error_map.cdf_img.copy_to_host(m_nerf.training.error_map.pmf_img_cpu);
		std::vector<float> cdf_img_cpu = m_nerf.training.error_map.pmf_img_cpu; // Copy unnormalized PDF into CDF buffer
		float cum = 0;
		for (float& f : cdf_img_cpu) {
			cum += f;
			f = cum;
		}
		float norm = 1.0f / cum;
		for (size_t i = 0; i < cdf_img_cpu.size(); ++i) {
			constexpr float MIN_PMF = 0.1f;
			m_nerf.training.error_map.pmf_img_cpu[i] = (1.0f - MIN_PMF) * m_nerf.training.error_map.pmf_img_cpu[i] * norm + MIN_PMF / (float)m_nerf.training.dataset.n_images;
			cdf_img_cpu[i] = (1.0f - MIN_PMF) * cdf_img_cpu[i] * norm + MIN_PMF * (float)(i+1) / (float)m_nerf.training.dataset.n_images;
		}
		m_nerf.training.error_map.cdf_img.copy_from_host(cdf_img_cpu);

		// Reset counters and decrease update rate.
		m_nerf.training.n_steps_since_error_map_update = 0;
		m_nerf.training.n_rays_since_error_map_update = 0;
		m_nerf.training.error_map.is_cdf_valid = true;

		m_nerf.training.n_steps_between_error_map_updates = (uint32_t)(m_nerf.training.n_steps_between_error_map_updates * 1.5f);
	}

	// Get extrinsics gradients
	m_nerf.training.n_steps_since_cam_update += n_training_steps;

	bool train_camera = m_nerf.training.optimize_extrinsics || m_nerf.training.optimize_distortion || m_nerf.training.optimize_focal_length || m_nerf.training.optimize_exposure;
	if (train_camera && m_nerf.training.n_steps_since_cam_update >= m_nerf.training.n_steps_between_cam_updates) {
		float per_camera_loss_scale = (float)m_nerf.training.n_images_for_training / LOSS_SCALE / (float)m_nerf.training.n_steps_between_cam_updates;

		if (m_nerf.training.optimize_extrinsics) {
			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_pos_gradient.data(), m_nerf.training.cam_pos_gradient_gpu.data(), m_nerf.training.cam_pos_gradient_gpu.get_bytes(), hipMemcpyDeviceToHost, stream));
			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_rot_gradient.data(), m_nerf.training.cam_rot_gradient_gpu.data(), m_nerf.training.cam_rot_gradient_gpu.get_bytes(), hipMemcpyDeviceToHost, stream));

			CUDA_CHECK_THROW(hipStreamSynchronize(stream));

			// Optimization step
			for (uint32_t i = 0; i < m_nerf.training.n_images_for_training; ++i) {
				Vector3f pos_gradient = m_nerf.training.cam_pos_gradient[i] * per_camera_loss_scale;
				Vector3f rot_gradient = m_nerf.training.cam_rot_gradient[i] * per_camera_loss_scale;

				float l2_reg = m_nerf.training.extrinsic_l2_reg;
				pos_gradient += m_nerf.training.cam_pos_offset[i].variable() * l2_reg;
				rot_gradient += m_nerf.training.cam_rot_offset[i].variable() * l2_reg;

				m_nerf.training.cam_pos_offset[i].set_learning_rate(std::max(1e-3f * std::pow(0.33f, (float)(m_nerf.training.cam_pos_offset[i].step() / 128)), m_optimizer->learning_rate()/1000.0f));
				m_nerf.training.cam_rot_offset[i].set_learning_rate(std::max(1e-3f * std::pow(0.33f, (float)(m_nerf.training.cam_rot_offset[i].step() / 128)), m_optimizer->learning_rate()/1000.0f));

				m_nerf.training.cam_pos_offset[i].step(pos_gradient);
				m_nerf.training.cam_rot_offset[i].step(rot_gradient);
			}

			m_nerf.training.update_transforms();
		}

		if (m_nerf.training.optimize_distortion) {
			linear_kernel(safe_divide, 0, stream,
				m_distortion.map->n_params(),
				m_distortion.map->gradients(),
				m_distortion.map->gradient_weights()
			);
			m_distortion.trainer->optimizer_step(stream, LOSS_SCALE*(float)m_nerf.training.n_steps_between_cam_updates);
		}

		if (m_nerf.training.optimize_focal_length) {
			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_focal_length_gradient.data(),m_nerf.training.cam_focal_length_gradient_gpu.data(),m_nerf.training.cam_focal_length_gradient_gpu.get_bytes(),hipMemcpyDeviceToHost, stream));
			CUDA_CHECK_THROW(hipStreamSynchronize(stream));
			Vector2f focal_length_gradient = m_nerf.training.cam_focal_length_gradient * per_camera_loss_scale;
			float l2_reg = m_nerf.training.intrinsic_l2_reg;
			focal_length_gradient += m_nerf.training.cam_focal_length_offset.variable() * l2_reg;
			m_nerf.training.cam_focal_length_offset.set_learning_rate(std::max(1e-3f * std::pow(0.33f, (float)(m_nerf.training.cam_focal_length_offset.step() / 128)),m_optimizer->learning_rate() / 1000.0f));
			m_nerf.training.cam_focal_length_offset.step(focal_length_gradient);
			m_nerf.training.update_metadata();
		}

		if (m_nerf.training.optimize_exposure) {
			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_exposure_gradient.data(), m_nerf.training.cam_exposure_gradient_gpu.data(), m_nerf.training.cam_exposure_gradient_gpu.get_bytes(), hipMemcpyDeviceToHost, stream));

			Array3f mean_exposure = Array3f::Constant(0.0f);

			// Optimization step
			for (uint32_t i = 0; i < m_nerf.training.n_images_for_training; ++i) {
				Array3f gradient = m_nerf.training.cam_exposure_gradient[i] * per_camera_loss_scale;

				float l2_reg = m_nerf.training.exposure_l2_reg;
				gradient += m_nerf.training.cam_exposure[i].variable() * l2_reg;

				m_nerf.training.cam_exposure[i].set_learning_rate(m_optimizer->learning_rate());
				m_nerf.training.cam_exposure[i].step(gradient);

				mean_exposure += m_nerf.training.cam_exposure[i].variable();
			}

			mean_exposure /= m_nerf.training.n_images_for_training;

			// Renormalize
			std::vector<Array3f> cam_exposures(m_nerf.training.n_images_for_training);
			for (uint32_t i = 0; i < m_nerf.training.n_images_for_training; ++i) {
				cam_exposures[i] = m_nerf.training.cam_exposure[i].variable() -= mean_exposure;
			}

			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_exposure_gpu.data(), cam_exposures.data(), m_nerf.training.cam_exposure_gpu.get_bytes(), hipMemcpyHostToDevice, stream));
		}

		m_nerf.training.n_steps_since_cam_update = 0;
	}
}

void Testbed::train_nerf_step_distill(uint32_t target_batch_size, uint32_t n_rays_per_batch, uint32_t* counter, uint32_t* compacted_counter, float* loss, hipStream_t stream) {
	int allowed_rays = n_rays_per_batch;
	n_rays_per_batch /= 2;
	int num_aux_rays = allowed_rays - n_rays_per_batch;

	const uint32_t padded_output_width = m_network->padded_output_width();
	const uint32_t max_samples = target_batch_size * 16; // Somewhat of a worst case
	const uint32_t floats_per_coord = sizeof(NerfCoordinate) / sizeof(float) + m_nerf_network->n_extra_dims();
	const uint32_t extra_stride = m_nerf_network->n_extra_dims() * sizeof(float); // extra stride on top of base NerfCoordinate struct

	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<
		uint32_t, // ray_indices
		Ray, // rays
		uint32_t, // numsteps
		float, // coords
		float, // max_level
		network_precision_t, // mlp_out
		network_precision_t, // dloss_dmlp_out
		float, // coords_compacted
		float, // coords_gradient
		float, // max_level_compacted
		uint32_t, // ray_counter
		Distiller*,
		int,
		AuxRay,
		AuxSample
	>(
		stream, &alloc,
		allowed_rays,
		allowed_rays,
		allowed_rays * 2,
		max_samples * floats_per_coord,
		max_samples,
		std::max(target_batch_size, max_samples) * padded_output_width,
		target_batch_size * padded_output_width,
		target_batch_size * floats_per_coord,
		target_batch_size * floats_per_coord,
		target_batch_size,
		1,
		m_nerf.tracer.edit_operators().size(),
		1,
		num_aux_rays,
		max_samples
		);

	// TODO: C++17 structured binding
	uint32_t* ray_indices = std::get<0>(scratch);
	Ray* rays = std::get<1>(scratch);
	uint32_t* numsteps = std::get<2>(scratch);
	float* coords = std::get<3>(scratch);
	float* max_level = std::get<4>(scratch);
	network_precision_t* mlp_out = std::get<5>(scratch);
	network_precision_t* dloss_dmlp_out = std::get<6>(scratch);
	float* coords_compacted = std::get<7>(scratch);
	float* coords_gradient = std::get<8>(scratch);
	float* max_level_compacted = std::get<9>(scratch);
	uint32_t* ray_counter = std::get<10>(scratch);
	Distiller** cage_infos = std::get<11>(scratch);
	int* to_duplicate = std::get<12>(scratch);
	AuxRay* auxrays = std::get<13>(scratch);
	AuxSample* auxsamples = std::get<14>(scratch);

	for (int i = 0; i < m_nerf.tracer.edit_operators().size(); i++)
	{
		auto gpuInfo = m_nerf.tracer.edit_operators()[i].get()->getDistiller();
		hipMemcpy(cage_infos + i, &gpuInfo, sizeof(Distiller*), hipMemcpyHostToDevice);
	}

	uint32_t max_inference;
	if (m_nerf.training.counters_rgb.measured_batch_size_before_compaction == 0) {
		m_nerf.training.counters_rgb.measured_batch_size_before_compaction = max_inference = max_samples;
	}
	else {
		max_inference = next_multiple(std::min(m_nerf.training.counters_rgb.measured_batch_size_before_compaction, max_samples), tcnn::batch_size_granularity);
	}

	GPUMatrix<float> coords_matrix((float*)coords, floats_per_coord, max_inference);
	GPUMatrix<network_precision_t> rgbsigma_matrix(mlp_out, padded_output_width, max_inference);

	GPUMatrix<float> compacted_coords_matrix((float*)coords_compacted, floats_per_coord, target_batch_size);
	GPUMatrix<network_precision_t> compacted_rgbsigma_matrix(mlp_out, padded_output_width, target_batch_size);

	GPUMatrix<network_precision_t> gradient_matrix(dloss_dmlp_out, padded_output_width, target_batch_size);

	if (m_training_step == 0) {
		m_nerf.training.counters_rgb.n_rays_total = 0;
	}

	uint32_t n_rays_total = m_nerf.training.counters_rgb.n_rays_total;
	m_nerf.training.counters_rgb.n_rays_total += n_rays_per_batch;
	m_nerf.training.n_rays_since_error_map_update += n_rays_per_batch;

	// If we have an envmap, prepare its gradient buffer
	float* envmap_gradient = m_nerf.training.train_envmap ? m_envmap.envmap->gradients() : nullptr;

	bool sample_focal_plane_proportional_to_error = m_nerf.training.error_map.is_cdf_valid && m_nerf.training.sample_focal_plane_proportional_to_error;
	bool sample_image_proportional_to_error = m_nerf.training.error_map.is_cdf_valid && m_nerf.training.sample_image_proportional_to_error;
	bool include_sharpness_in_error = m_nerf.training.include_sharpness_in_error;
	// This is low-overhead enough to warrant always being on.
	// It makes for useful visualizations of the training error.
	bool accumulate_error = true;

	CUDA_CHECK_THROW(hipMemsetAsync(ray_counter, 0, sizeof(uint32_t), stream));

	hipMemset(auxsamples, 0, max_samples * sizeof(AuxSample));
	hipMemset(to_duplicate, 0, sizeof(int));

	linear_kernel(generate_training_samples_nerf_distill, 0, stream,
		n_rays_per_batch,
		m_aabb,
		max_inference,
		n_rays_total,
		m_rng,
		m_nerf.training.dataset.rays_data.data(),
		ray_counter,
		counter,
		ray_indices,
		rays,
		numsteps,
		PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords, 1, 0, extra_stride),
		m_nerf.training.image_resolution,
		m_nerf.training.n_images_for_training,
		m_nerf.training.metadata_gpu.data(),
		m_nerf.training.transforms_gpu.data(),
		m_nerf.density_grid_bitfield.data(),
		m_max_level_rand_training,
		max_level,
		m_nerf.training.snap_to_pixel_centers,
		m_nerf.training.train_envmap,
		m_nerf.cone_angle_constant,
		m_distortion.map->params(),
		m_distortion.resolution,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_x_cond_y.data() : nullptr,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_y.data() : nullptr,
		sample_image_proportional_to_error ? m_nerf.training.error_map.cdf_img.data() : nullptr,
		m_nerf.training.error_map.cdf_resolution,
		m_nerf.training.dataset.images_data.data(),
		cage_infos,
		m_nerf.tracer.edit_operators().size(),
		to_duplicate,
		auxrays,
		num_aux_rays,
		auxsamples
	);

	int check_count;
	hipMemcpy(&check_count, to_duplicate, sizeof(int), hipMemcpyDeviceToHost);
	check_count = min(check_count, num_aux_rays);

	if (check_count > 0)
	{
		linear_kernel(generate_extra_training_samples_nerf_distill, 0, stream,
			check_count,
			m_aabb,
			max_inference,
			ray_counter,
			counter,
			ray_indices,
			rays,
			numsteps,
			PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords, 1, 0, extra_stride),
			m_nerf.density_grid_bitfield.data(),
			m_max_level_rand_training,
			max_level,
			m_nerf.training.train_envmap,
			cage_infos,
			m_nerf.tracer.edit_operators().size(),
			auxrays,
			auxsamples
		);
	}

	auto hg_enc = dynamic_cast<GridEncoding<network_precision_t>*>(m_encoding.get());
	if (hg_enc) {
		hg_enc->set_max_level_gpu(m_max_level_rand_training ? max_level : nullptr);
	}

	m_network->inference_mixed_precision(stream, coords_matrix, rgbsigma_matrix, false);

	if (hg_enc) {
		hg_enc->set_max_level_gpu(m_max_level_rand_training ? max_level_compacted : nullptr);
	}

	linear_kernel(compute_loss_kernel_train_nerf_distill, 0, stream,
		allowed_rays,
		n_rays_per_batch,
		m_aabb,
		n_rays_total,
		m_rng,
		target_batch_size,
		ray_counter,
		LOSS_SCALE,
		padded_output_width,
		m_envmap.envmap->params(),
		envmap_gradient,
		m_envmap.resolution,
		m_envmap.loss_type,
		m_background_color.head<3>(),
		m_color_space,
		m_nerf.training.random_bg_color,
		m_nerf.training.linear_colors,
		m_nerf.training.dataset.images_data.data(),
		m_nerf.training.n_images_for_training,
		m_nerf.training.image_resolution,
		mlp_out,
		compacted_counter,
		ray_indices,
		rays,
		numsteps,
		PitchedPtr<const NerfCoordinate>((NerfCoordinate*)coords, 1, 0, extra_stride),
		PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords_compacted, 1, 0, extra_stride),
		dloss_dmlp_out,
		m_nerf.training.loss_type,
		loss,
		m_max_level_rand_training,
		max_level_compacted,
		m_nerf.rgb_activation,
		m_nerf.density_activation,
		m_nerf.training.snap_to_pixel_centers,
		accumulate_error ? m_nerf.training.error_map.data.data() : nullptr,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_x_cond_y.data() : nullptr,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_y.data() : nullptr,
		sample_image_proportional_to_error ? m_nerf.training.error_map.cdf_img.data() : nullptr,
		m_nerf.training.error_map.resolution,
		m_nerf.training.error_map.cdf_resolution,
		include_sharpness_in_error ? m_nerf.training.dataset.sharpness_data.data() : nullptr,
		m_nerf.training.dataset.sharpness_resolution,
		m_nerf.training.sharpness_grid.data(),
		m_nerf.density_grid.data(),
		m_nerf.density_grid_mean.data(),
		m_nerf.training.cam_exposure_gpu.data(),
		m_nerf.training.optimize_exposure ? m_nerf.training.cam_exposure_gradient_gpu.data() : nullptr,
		m_nerf.training.near_distance,
		cage_infos,
		auxsamples
	);

	fill_rollover_and_rescale<network_precision_t> << <n_blocks_linear(target_batch_size * padded_output_width), n_threads_linear, 0, stream >> > (
		target_batch_size, padded_output_width, compacted_counter, dloss_dmlp_out
		);
	fill_rollover<float> << <n_blocks_linear(target_batch_size * floats_per_coord), n_threads_linear, 0, stream >> > (
		target_batch_size, floats_per_coord, compacted_counter, (float*)coords_compacted
		);
	fill_rollover<float> << <n_blocks_linear(target_batch_size), n_threads_linear, 0, stream >> > (
		target_batch_size, 1, compacted_counter, max_level_compacted
		);

	bool train_camera = m_nerf.training.optimize_extrinsics || m_nerf.training.optimize_distortion || m_nerf.training.optimize_focal_length;
	GPUMatrix<float> coords_gradient_matrix((float*)coords_gradient, floats_per_coord, target_batch_size);

	{
		auto ctx = m_network->forward(stream, compacted_coords_matrix, &compacted_rgbsigma_matrix, false, train_camera);
		m_network->backward(stream, *ctx, compacted_coords_matrix, compacted_rgbsigma_matrix, gradient_matrix, train_camera ? &coords_gradient_matrix : nullptr, false, EGradientMode::Overwrite);
	}

	if (train_camera) {
		// Compute camera gradients
		linear_kernel(compute_cam_gradient_train_nerf, 0, stream,
			n_rays_per_batch,
			n_rays_total,
			m_rng,
			m_aabb,
			ray_counter,
			m_nerf.training.transforms_gpu.data(),
			m_nerf.training.image_resolution,
			m_nerf.training.snap_to_pixel_centers,
			m_nerf.training.optimize_extrinsics ? m_nerf.training.cam_pos_gradient_gpu.data() : nullptr,
			m_nerf.training.optimize_extrinsics ? m_nerf.training.cam_rot_gradient_gpu.data() : nullptr,
			m_nerf.training.n_images_for_training,
			ray_indices,
			rays,
			numsteps,
			PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords_compacted, 1, 0, extra_stride),
			PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords_gradient, 1, 0, extra_stride),
			m_nerf.training.optimize_distortion ? m_distortion.map->gradients() : nullptr,
			m_nerf.training.optimize_distortion ? m_distortion.map->gradient_weights() : nullptr,
			m_distortion.resolution,
			m_nerf.training.optimize_focal_length ? m_nerf.training.cam_focal_length_gradient_gpu.data() : nullptr,
			sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_x_cond_y.data() : nullptr,
			sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_y.data() : nullptr,
			sample_image_proportional_to_error ? m_nerf.training.error_map.cdf_img.data() : nullptr,
			m_nerf.training.error_map.cdf_resolution
		);
	}

	m_rng.advance();

	if (hg_enc) {
		hg_enc->set_max_level_gpu(nullptr);
	}
}

void Testbed::train_nerf_step(uint32_t target_batch_size, uint32_t n_rays_per_batch, uint32_t* counter, uint32_t* compacted_counter, float* loss, hipStream_t stream) {
	const uint32_t padded_output_width = m_network->padded_output_width();
	const uint32_t max_samples = target_batch_size * 16; // Somewhat of a worst case
	const uint32_t floats_per_coord = sizeof(NerfCoordinate) / sizeof(float) + m_nerf_network->n_extra_dims();
	const uint32_t extra_stride = m_nerf_network->n_extra_dims() * sizeof(float); // extra stride on top of base NerfCoordinate struct

	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<
		uint32_t, // ray_indices
		Ray, // rays
		uint32_t, // numsteps
		float, // coords
		float, // max_level
		network_precision_t, // mlp_out
		network_precision_t, // dloss_dmlp_out
		float, // coords_compacted
		float, // coords_gradient
		float, // max_level_compacted
		uint32_t // ray_counter
	>(
		stream, &alloc,
		n_rays_per_batch,
		n_rays_per_batch,
		n_rays_per_batch * 2,
		max_samples * floats_per_coord,
		max_samples,
		std::max(target_batch_size, max_samples) * padded_output_width,
		target_batch_size * padded_output_width,
		target_batch_size * floats_per_coord,
		target_batch_size * floats_per_coord,
		target_batch_size,
		1
	);

	// TODO: C++17 structured binding
	uint32_t* ray_indices = std::get<0>(scratch);
	Ray* rays = std::get<1>(scratch);
	uint32_t* numsteps = std::get<2>(scratch);
	float* coords = std::get<3>(scratch);
	float* max_level = std::get<4>(scratch);
	network_precision_t* mlp_out = std::get<5>(scratch);
	network_precision_t* dloss_dmlp_out = std::get<6>(scratch);
	float* coords_compacted = std::get<7>(scratch);
	float* coords_gradient = std::get<8>(scratch);
	float* max_level_compacted = std::get<9>(scratch);
	uint32_t* ray_counter = std::get<10>(scratch);

	uint32_t max_inference;
	if (m_nerf.training.counters_rgb.measured_batch_size_before_compaction == 0) {
		m_nerf.training.counters_rgb.measured_batch_size_before_compaction = max_inference = max_samples;
	} else {
		max_inference = next_multiple(std::min(m_nerf.training.counters_rgb.measured_batch_size_before_compaction, max_samples), tcnn::batch_size_granularity);
	}

	GPUMatrix<float> coords_matrix((float*)coords, floats_per_coord, max_inference);
	GPUMatrix<network_precision_t> rgbsigma_matrix(mlp_out, padded_output_width, max_inference);

	GPUMatrix<float> compacted_coords_matrix((float*)coords_compacted, floats_per_coord, target_batch_size);
	GPUMatrix<network_precision_t> compacted_rgbsigma_matrix(mlp_out, padded_output_width, target_batch_size);

	GPUMatrix<network_precision_t> gradient_matrix(dloss_dmlp_out, padded_output_width, target_batch_size);

	if (m_training_step == 0) {
		m_nerf.training.counters_rgb.n_rays_total = 0;
	}

	uint32_t n_rays_total = m_nerf.training.counters_rgb.n_rays_total;
	m_nerf.training.counters_rgb.n_rays_total += n_rays_per_batch;
	m_nerf.training.n_rays_since_error_map_update += n_rays_per_batch;

	// If we have an envmap, prepare its gradient buffer
	float* envmap_gradient = m_nerf.training.train_envmap ? m_envmap.envmap->gradients() : nullptr;

	bool sample_focal_plane_proportional_to_error = m_nerf.training.error_map.is_cdf_valid && m_nerf.training.sample_focal_plane_proportional_to_error;
	bool sample_image_proportional_to_error = m_nerf.training.error_map.is_cdf_valid && m_nerf.training.sample_image_proportional_to_error;
	bool include_sharpness_in_error = m_nerf.training.include_sharpness_in_error;
	// This is low-overhead enough to warrant always being on.
	// It makes for useful visualizations of the training error.
	bool accumulate_error = true;

	CUDA_CHECK_THROW(hipMemsetAsync(ray_counter, 0, sizeof(uint32_t), stream));

	linear_kernel(generate_training_samples_nerf, 0, stream,
		n_rays_per_batch,
		m_aabb,
		max_inference,
		n_rays_total,
		m_rng,
		m_nerf.training.dataset.rays_data.data(),
		ray_counter,
		counter,
		ray_indices,
		rays,
		numsteps,
		PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords, 1, 0, extra_stride),
		m_nerf.training.image_resolution,
		m_nerf.training.n_images_for_training,
		m_nerf.training.metadata_gpu.data(),
		m_nerf.training.transforms_gpu.data(),
		m_nerf.density_grid_bitfield.data(),
		m_max_level_rand_training,
		max_level,
		m_nerf.training.snap_to_pixel_centers,
		m_nerf.training.train_envmap,
		m_nerf.cone_angle_constant,
		m_distortion.map->params(),
		m_distortion.resolution,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_x_cond_y.data() : nullptr,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_y.data() : nullptr,
		sample_image_proportional_to_error ? m_nerf.training.error_map.cdf_img.data() : nullptr,
		m_nerf.training.error_map.cdf_resolution,
		m_nerf.training.dataset.images_data.data()
	);

	auto hg_enc = dynamic_cast<GridEncoding<network_precision_t>*>(m_encoding.get());
	if (hg_enc) {
		hg_enc->set_max_level_gpu(m_max_level_rand_training ? max_level : nullptr);
	}

	m_network->inference_mixed_precision(stream, coords_matrix, rgbsigma_matrix, false);

	if (hg_enc) {
		hg_enc->set_max_level_gpu(m_max_level_rand_training ? max_level_compacted : nullptr);
	}

	linear_kernel(compute_loss_kernel_train_nerf, 0, stream,
		n_rays_per_batch,
		m_aabb,
		n_rays_total,
		m_rng,
		target_batch_size,
		ray_counter,
		LOSS_SCALE,
		padded_output_width,
		m_envmap.envmap->params(),
		envmap_gradient,
		m_envmap.resolution,
		m_envmap.loss_type,
		m_background_color.head<3>(),
		m_color_space,
		m_nerf.training.random_bg_color,
		m_nerf.training.linear_colors,
		m_nerf.training.dataset.images_data.data(),
		m_nerf.training.n_images_for_training,
		m_nerf.training.image_resolution,
		mlp_out,
		compacted_counter,
		ray_indices,
		rays,
		numsteps,
		PitchedPtr<const NerfCoordinate>((NerfCoordinate*)coords, 1, 0, extra_stride),
		PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords_compacted, 1 ,0, extra_stride),
		dloss_dmlp_out,
		m_nerf.training.loss_type,
		loss,
		m_max_level_rand_training,
		max_level_compacted,
		m_nerf.rgb_activation,
		m_nerf.density_activation,
		m_nerf.training.snap_to_pixel_centers,
		accumulate_error ? m_nerf.training.error_map.data.data() : nullptr,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_x_cond_y.data() : nullptr,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_y.data() : nullptr,
		sample_image_proportional_to_error ? m_nerf.training.error_map.cdf_img.data() : nullptr,
		m_nerf.training.error_map.resolution,
		m_nerf.training.error_map.cdf_resolution,
		include_sharpness_in_error ? m_nerf.training.dataset.sharpness_data.data() : nullptr,
		m_nerf.training.dataset.sharpness_resolution,
		m_nerf.training.sharpness_grid.data(),
		m_nerf.density_grid.data(),
		m_nerf.density_grid_mean.data(),
		m_nerf.training.cam_exposure_gpu.data(),
		m_nerf.training.optimize_exposure ? m_nerf.training.cam_exposure_gradient_gpu.data() : nullptr,
		m_nerf.training.near_distance
	);

	fill_rollover_and_rescale<network_precision_t><<<n_blocks_linear(target_batch_size*padded_output_width), n_threads_linear, 0, stream>>>(
		target_batch_size, padded_output_width, compacted_counter, dloss_dmlp_out
	);
	fill_rollover<float><<<n_blocks_linear(target_batch_size * floats_per_coord), n_threads_linear, 0, stream>>>(
		target_batch_size, floats_per_coord, compacted_counter, (float*)coords_compacted
	);
	fill_rollover<float><<<n_blocks_linear(target_batch_size), n_threads_linear, 0, stream>>>(
		target_batch_size, 1, compacted_counter, max_level_compacted
	);

	bool train_camera = m_nerf.training.optimize_extrinsics || m_nerf.training.optimize_distortion || m_nerf.training.optimize_focal_length;
	GPUMatrix<float> coords_gradient_matrix((float*)coords_gradient, floats_per_coord, target_batch_size);

	{
		auto ctx = m_network->forward(stream, compacted_coords_matrix, &compacted_rgbsigma_matrix, false, train_camera);
		m_network->backward(stream, *ctx, compacted_coords_matrix, compacted_rgbsigma_matrix, gradient_matrix, train_camera ? &coords_gradient_matrix : nullptr, false, EGradientMode::Overwrite);
	}

	if (train_camera) {
		// Compute camera gradients
		linear_kernel(compute_cam_gradient_train_nerf, 0, stream,
			n_rays_per_batch,
			n_rays_total,
			m_rng,
			m_aabb,
			ray_counter,
			m_nerf.training.transforms_gpu.data(),
			m_nerf.training.image_resolution,
			m_nerf.training.snap_to_pixel_centers,
			m_nerf.training.optimize_extrinsics ? m_nerf.training.cam_pos_gradient_gpu.data() : nullptr,
			m_nerf.training.optimize_extrinsics ? m_nerf.training.cam_rot_gradient_gpu.data() : nullptr,
			m_nerf.training.n_images_for_training,
			ray_indices,
			rays,
			numsteps,
			PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords_compacted, 1, 0, extra_stride),
			PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords_gradient, 1, 0, extra_stride),
			m_nerf.training.optimize_distortion ? m_distortion.map->gradients() : nullptr,
			m_nerf.training.optimize_distortion ? m_distortion.map->gradient_weights() : nullptr,
			m_distortion.resolution,
			m_nerf.training.optimize_focal_length ? m_nerf.training.cam_focal_length_gradient_gpu.data() : nullptr,
			sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_x_cond_y.data() : nullptr,
			sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_y.data() : nullptr,
			sample_image_proportional_to_error ? m_nerf.training.error_map.cdf_img.data() : nullptr,
			m_nerf.training.error_map.cdf_resolution
		);
	}

	m_rng.advance();

	if (hg_enc) {
		hg_enc->set_max_level_gpu(nullptr);
	}
}

void Testbed::reset_density_grid_nerf(hipStream_t stream) {
	const uint32_t n_elements = NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_CASCADES();
	// Don't forget to allocate data before setting it!
	m_nerf.density_grid.enlarge(n_elements);
	CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.density_grid.data(), 0, sizeof(float)*n_elements, stream));
}

void Testbed::update_density_grid_nerf_3d(bool initialize, hipStream_t stream) {
	float alpha = m_nerf.training.density_grid_decay;
	uint32_t n_cascades = m_nerf.max_cascade+1;
	m_training_step = 1;
	if (initialize) {
		update_density_grid_nerf(alpha, NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE()*n_cascades, 0, stream);
	} else {
		update_density_grid_nerf(alpha, NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE()/4*n_cascades, NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE()/4*n_cascades, stream);
	}
}

void Testbed::training_prep_nerf(uint32_t batch_size, uint32_t n_training_steps, hipStream_t stream) {
	float alpha = std::pow(m_nerf.training.density_grid_decay, n_training_steps / 16.0f);
	uint32_t n_cascades = m_nerf.max_cascade+1;
	if (m_training_step < 256) {
		update_density_grid_nerf(alpha, NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE()*n_cascades, 0, stream);
	} else {
		update_density_grid_nerf(alpha, NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE()/4*n_cascades, NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE()/4*n_cascades, stream);
	}
}

void Testbed::optimise_mesh_step(uint32_t n_steps) {
	uint32_t n_verts = (uint32_t)m_mesh.verts.size();
	if (!n_verts) {
		return;
	}

	const uint32_t padded_output_width = m_nerf_network->padded_density_output_width();
	const uint32_t floats_per_coord = sizeof(NerfCoordinate) / sizeof(float) + m_nerf_network->n_extra_dims();
	const uint32_t extra_stride = m_nerf_network->n_extra_dims() * sizeof(float);
	GPUMemory<float> coords(n_verts * floats_per_coord);
	GPUMemory<network_precision_t> mlp_out(n_verts * padded_output_width);

	GPUMatrix<float> positions_matrix((float*)coords.data(), floats_per_coord, n_verts);
	GPUMatrix<network_precision_t, RM> density_matrix(mlp_out.data(), padded_output_width, n_verts);

	for (uint32_t i = 0; i < n_steps; ++i) {
		linear_kernel(generate_nerf_network_inputs_from_positions, 0, m_inference_stream, n_verts, m_aabb, m_mesh.verts.data(), PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords.data(), 1, 0, extra_stride), m_nerf.light_dir.normalized());

		// For each optimizer step, we need the density at the given pos...
		m_nerf_network->density(m_inference_stream, positions_matrix, density_matrix);
		// ...as well as the input gradient w.r.t. density, which we will store in the nerf coords.
		m_nerf_network->input_gradient(m_inference_stream, 3, positions_matrix, positions_matrix);
		// and the 1ring centroid for laplacian smoothing
		compute_mesh_1ring(m_mesh.verts, m_mesh.indices, m_mesh.verts_smoothed, m_mesh.vert_normals);

		// With these, we can compute a gradient that points towards the threshold-crossing of density...
		compute_mesh_opt_gradients(
			m_mesh.thresh,
			m_mesh.verts,
			m_mesh.vert_normals,
			m_mesh.verts_smoothed,
			mlp_out.data(),
			floats_per_coord,
			(const float*)coords.data(),
			m_mesh.verts_gradient,
			m_mesh.smooth_amount,
			m_mesh.density_amount,
			m_mesh.inflate_amount
		);

		// ...that we can pass to the optimizer.
		m_mesh.verts_optimizer->step(m_inference_stream, 1.0f, (float*)m_mesh.verts.data(), (float*)m_mesh.verts.data(), (float*)m_mesh.verts_gradient.data());
	}
}

void Testbed::compute_mesh_vertex_colors() {
	uint32_t n_verts = (uint32_t)m_mesh.verts.size();
	if (!n_verts) {
		return;
	}

	m_mesh.vert_colors.resize(n_verts);
	m_mesh.vert_colors.memset(0);

	if (m_testbed_mode == ETestbedMode::Nerf) {
		const uint32_t floats_per_coord = sizeof(NerfCoordinate) / sizeof(float) + m_nerf_network->n_extra_dims();
		const uint32_t extra_stride = m_nerf_network->n_extra_dims() * sizeof(float);
		GPUMemory<float> coords(n_verts * floats_per_coord);
		GPUMemory<float> mlp_out(n_verts * 4);

		GPUMatrix<float> positions_matrix((float*)coords.data(), floats_per_coord, n_verts);
		GPUMatrix<float> color_matrix(mlp_out.data(), 4, n_verts);
		linear_kernel(generate_nerf_network_inputs_from_positions, 0, m_inference_stream, n_verts, m_aabb, m_mesh.verts.data(), PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords.data(), 1, 0, extra_stride), m_nerf.light_dir.normalized());
		m_network->inference(m_inference_stream, positions_matrix, color_matrix);
		linear_kernel(extract_srgb_with_activation, 0, m_inference_stream, n_verts * 3, 3, mlp_out.data(), (float*)m_mesh.vert_colors.data(), m_nerf.rgb_activation, m_nerf.training.linear_colors);
	}
}

GPUMemory<float> Testbed::get_density_on_grid(Vector3i res3d, const BoundingBox& aabb) {
	const uint32_t n_elements = (res3d.x()*res3d.y()*res3d.z());
	GPUMemory<float> density(n_elements);

	const uint32_t batch_size = std::min(n_elements, 1u<<20);
	bool nerf_mode = m_testbed_mode == ETestbedMode::Nerf;

	const uint32_t padded_output_width = nerf_mode ? m_nerf_network->padded_density_output_width() : m_network->padded_output_width();

	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<
		NerfPosition,
		network_precision_t
	>(m_inference_stream, &alloc, n_elements, batch_size * padded_output_width);

	NerfPosition* positions = std::get<0>(scratch);
	network_precision_t* mlp_out = std::get<1>(scratch);

	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)res3d.x(), threads.x), div_round_up((uint32_t)res3d.y(), threads.y), div_round_up((uint32_t)res3d.z(), threads.z) };

	BoundingBox unit_cube = BoundingBox{Vector3f::Zero(), Vector3f::Ones()};
	generate_grid_samples_nerf_uniform<<<blocks, threads, 0, m_inference_stream>>>(res3d, m_nerf.density_grid_ema_step, aabb, nerf_mode ? m_aabb : unit_cube , positions);

	// Only process 1m elements at a time
	for (uint32_t offset = 0; offset < n_elements; offset += batch_size) {
		uint32_t local_batch_size = std::min(n_elements - offset, batch_size);

		GPUMatrix<network_precision_t, RM> density_matrix(mlp_out, padded_output_width, local_batch_size);

		GPUMatrix<float> positions_matrix((float*)(positions + offset), sizeof(NerfPosition)/sizeof(float), local_batch_size);
		if (nerf_mode) {
			m_nerf_network->density(m_inference_stream, positions_matrix, density_matrix);
		} else {
			m_network->inference_mixed_precision(m_inference_stream, positions_matrix, density_matrix);
		}
		linear_kernel(grid_samples_half_to_float, 0, m_inference_stream,
			local_batch_size,
			m_aabb,
			density.data() + offset , //+ axis_step * n_elements,
			mlp_out,
			m_nerf.density_activation,
			positions + offset,
			nerf_mode ? m_nerf.density_grid.data() : nullptr
		);
	}

	return density;
}

GPUMemory<Eigen::Array4f> Testbed::get_rgba_on_grid(Vector3i res3d, Eigen::Vector3f ray_dir) {
	const uint32_t n_elements = (res3d.x()*res3d.y()*res3d.z());
	GPUMemory<Eigen::Array4f> rgba(n_elements);
	GPUMemory<NerfCoordinate> positions(n_elements);
	const uint32_t batch_size = std::min(n_elements, 1u<<20);

	// generate inputs
	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)res3d.x(), threads.x), div_round_up((uint32_t)res3d.y(), threads.y), div_round_up((uint32_t)res3d.z(), threads.z) };
	generate_grid_samples_nerf_uniform_dir<<<blocks, threads, 0, m_inference_stream>>>(res3d, m_nerf.density_grid_ema_step, m_render_aabb, m_aabb, ray_dir, positions.data());

	// Only process 1m elements at a time
	for (uint32_t offset = 0; offset < n_elements; offset += batch_size) {
		uint32_t local_batch_size = std::min(n_elements - offset, batch_size);

		// run network
		GPUMatrix<float> positions_matrix((float*) (positions.data() + offset), sizeof(NerfCoordinate)/sizeof(float), local_batch_size);
		GPUMatrix<float> rgbsigma_matrix((float*) (rgba.data() + offset), 4, local_batch_size);
		m_network->inference(m_inference_stream, positions_matrix, rgbsigma_matrix);

		// convert network output to RGBA (in place)
		linear_kernel(compute_nerf_density, 0, m_inference_stream, local_batch_size, rgba.data() + offset, m_nerf.rgb_activation, m_nerf.density_activation);
	}
	return rgba;
}

int Testbed::marching_cubes(Vector3i res3d, const BoundingBox& aabb, float thresh) {
	res3d.x() = next_multiple((unsigned int)res3d.x(), 16u);
	res3d.y() = next_multiple((unsigned int)res3d.y(), 16u);
	res3d.z() = next_multiple((unsigned int)res3d.z(), 16u);

	if (thresh == std::numeric_limits<float>::max()) {
		thresh = m_mesh.thresh;
	}

	GPUMemory<float> density = get_density_on_grid(res3d, aabb);
	marching_cubes_gpu(m_inference_stream, aabb, res3d, thresh, density, m_mesh.verts, m_mesh.indices);

	uint32_t n_verts = (uint32_t)m_mesh.verts.size();
	m_mesh.verts_gradient.resize(n_verts);

	m_mesh.trainable_verts = std::make_shared<TrainableBuffer<3, 1, float>>(Matrix<int, 1, 1>{(int)n_verts});
	m_mesh.verts_gradient.copy_from_device(m_mesh.verts); // Make sure the vertices don't get destroyed in the initialization

	pcg32 rnd{m_seed};
	m_mesh.trainable_verts->initialize_params(rnd, (float*)m_mesh.verts.data(), (float*)m_mesh.verts.data(), (float*)m_mesh.verts.data(), (float*)m_mesh.verts.data(), (float*)m_mesh.verts_gradient.data());
	m_mesh.verts.copy_from_device(m_mesh.verts_gradient);

	m_mesh.verts_optimizer.reset(create_optimizer<float>({
		{"otype", "Adam"},
		{"learning_rate", 1e-4},
		{"beta1", 0.9f},
		{"beta2", 0.99f},
	}));

	m_mesh.verts_optimizer->allocate(m_mesh.trainable_verts);

	compute_mesh_1ring(m_mesh.verts, m_mesh.indices, m_mesh.verts_smoothed, m_mesh.vert_normals);
	compute_mesh_vertex_colors();

	return (int)(m_mesh.indices.size()/3);
}

uint8_t* Testbed::Nerf::get_density_grid_bitfield_mip(uint32_t mip) {
	return density_grid_bitfield.data() + grid_mip_offset(mip)/8;
}

int Testbed::find_best_training_view(int default_view) {
	int bestimage = default_view;
	float bestscore = 1000.f;
	for (int i = 0; i < m_nerf.training.n_images_for_training; ++i) {
		float score = (m_nerf.training.dataset.xforms[i].start.col(3) - m_camera.col(3)).norm();
		score += 0.25f * (m_nerf.training.dataset.xforms[i].start.col(2) - m_camera.col(2)).norm();
		if (score < bestscore) { bestscore = score; bestimage = i; }
	}
	return bestimage;
}

NGP_NAMESPACE_END
